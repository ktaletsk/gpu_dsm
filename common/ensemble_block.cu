#include "hip/hip_runtime.h"
// Copyright 2015 Marat Andreev, Konstantin Taletskiy, Maria Katzarova
// 
// This file is part of gpu_dsm.
// 
// gpu_dsm is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// at your option) any later version.
// 
// gpu_dsm is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with gpu_dsm.  If not, see <http://www.gnu.org/licenses/>.
#include "textures_surfaces.h"
#include "chain.h"
#include "gpu_random.h"
#include "ensemble_kernel.cu"
#include "ensemble_block.h"
#include "correlator.h"
#include <vector>
#define max_sync_interval 1E5
//variable arrays, that are common for all the blocks
gpu_Ran *d_random_gens; // device random number generators
gpu_Ran *d_random_gens2; //first is used to pick jump process, second is used for creation of new entanglements
//temporary arrays fpr random numbers sequences
hipArray* d_uniformrand; // uniform random number supply //used to pick jump process
hipArray* d_taucd_gauss_rand_CD; // 1x uniform + 3x normal distributed random number supply// used for creating entanglements by SD
hipArray* d_taucd_gauss_rand_SD; // used for creating entanglements by SD
int steps_count = 0;    //time step count
int *d_tau_CD_used_SD;
int *d_tau_CD_used_CD;
int *d_rand_used;
//std::vector<float> pcd_data;
hipArray* d_a_QN; //device arrays for vector part of chain conformations
hipArray* d_a_tCD; // these arrays used by time evolution kernels
hipArray* d_b_QN;
hipArray* d_b_tCD;
hipArray* d_a_R1;
hipArray* d_b_R1;
hipArray* d_corr_a;
hipArray* d_corr_b;

hipArray* d_sum_W; // sum of probabilities for each entanglement
hipArray* d_stress; // stress calculation temp array

// There are two arrays with the vector part of the chain conformations  on device.
// And there is only one array with the scalar part of the chain conformations
// Every timestep the vector part is copied from one array to another.
// The coping is done in entanglement parallel portion of the code
// This allows to use textures/surfaces and speeds up memory access
// The scalar part(chain headers) is updated in the chain parallel portion of the code
// Chain headers occupy less memory,and there are no specific memory access technics for them.

void ensemble_block::init(int nc_, vector_chains chains_, scalar_chains* chain_heads_, int nsteps_){
	//allocates arrays, copies chain conformations to device
	//ensemble_call_block *cb pointer for call block structure, just ref parameter
	//int nc  is a number of chains in this ensemble_call_block.
	//sstrentp chains, chain_head* chain_heads pointers to array with the chain conformations

	nc = nc_;
	nsteps = nsteps_;
	chains = chains_;
	chain_heads = chain_heads_;
	block_time = universal_time;

	//blank dynamics probabalities
	float *buffer = new float[z_max * nc];
	memset(buffer, 0, sizeof(float) * z_max * nc);
	hipMemcpy2DToArray(d_sum_W, 0, 0, buffer, z_max * sizeof(float), z_max * sizeof(float), nc, hipMemcpyHostToDevice);
	delete[] buffer;

	hipMalloc((void**) &gpu_chain_heads, sizeof(scalar_chains) * nc);
	hipMemcpy(gpu_chain_heads, chain_heads, sizeof(scalar_chains) * nc, hipMemcpyHostToDevice);

	// allocating device arrays
	hipMalloc(&d_dt, sizeof(float) * nc);
	hipMemset(d_dt, 0, sizeof(float) * nc);

	hipMalloc(&d_offset, sizeof(int) * nc);
	hipMemset(d_offset, 0xff, sizeof(float) * nc);

	hipMalloc(&reach_flag, sizeof(float) * nc);
	hipMalloc(&d_new_strent, sizeof(float) * 4 * nc);
	hipMalloc(&d_new_tau_CD, sizeof(float) * nc);

	int s = ceil(log((float)nsteps/correlator_size)/log(correlator_res)) + 1; //number of correlator levels
	cout << "number of correlator levels" << '\t' << s << '\n' << '\n';
	corr = new correlator(nc, s);//Allocate memory for c_correlator structure in cb
	hipMalloc((void**) &d_write_time, sizeof(int) * nc);//Allocated memory on device for correlator time for every chain in block
	hipMemset(d_write_time, 0, sizeof(int) * nc);	//Initialize d_correlator_time with zeros

	hipMallocManaged((void**)&(stress_average), sizeof(float4) * nsteps);
	CUT_CHECK_ERROR("kernel execution failed");
}

template<int type> int  ensemble_block::time_step(double reach_time, int correlator_type, bool* run_flag, int *progress_bar) {
	//bind textures/surfaces, perform time evolution, unbind textures/surfaces

	//Declare and create streams for parallel correlator update
	hipStream_t stream_calc;
	hipStream_t stream_update;
	hipStreamCreate(&stream_calc);
	hipStreamCreate(&stream_update);

	hipChannelFormatDesc channelDesc1 = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat); //to read float4
	hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat); //to read float

	//loop preparing
	dim3 dimBlock(tpb_strent_kernel, tpb_strent_kernel);
	dim3 dimGrid((z_max + dimBlock.x - 1) / dimBlock.x, (nc + dimBlock.y - 1) / dimBlock.y);

	steps_count = 0;
	activate_block();

	float time_step_interval = reach_time - block_time;
	int number_of_syncs = int(floor((time_step_interval - 0.5) / max_sync_interval)) + 1;

	float *rtbuffer;
	hipHostMalloc(&rtbuffer, sizeof(float)*nc);

	int *tbuffer;
	hipHostMalloc(&tbuffer, sizeof(int)*nc);

	bool texture_flag = true;
	//Loop begins
	for (int i_sync = 0; i_sync < number_of_syncs; i_sync++) {
		float sync_interval = max_sync_interval;
		if ((i_sync + 1) == number_of_syncs)
			sync_interval = time_step_interval - i_sync * max_sync_interval;

		bool reach_flag_all = false;
		hipMemset(reach_flag, 0, sizeof(float) * nc);

		//update universal_time on device
		float tf = block_time + i_sync * max_sync_interval;
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_universal_time), &tf, sizeof(float)));

		while (!reach_flag_all) {
			if (!(steps_count & 0x00000001)) { //For odd number of steps

				hipBindTextureToArray(t_a_QN, d_a_QN, channelDesc4);
				cudaBindSurfaceToArray(s_b_QN, d_b_QN);
				hipBindTextureToArray(t_a_tCD, d_a_tCD, channelDesc1);
				cudaBindSurfaceToArray(s_b_tCD, d_b_tCD);
				hipBindTextureToArray(t_a_R1, d_a_R1, channelDesc4);
				cudaBindSurfaceToArray(s_b_R1, d_b_R1);
				if (texture_flag == true){
					cudaBindSurfaceToArray(s_corr, d_corr_b);
				} else {
					cudaBindSurfaceToArray(s_corr, d_corr_a);
				}

				strent_kernel<type><<<dimGrid, dimBlock,0,stream_calc>>>(gpu_chain_heads, d_dt, d_offset, d_new_strent, d_new_tau_CD);
				CUT_CHECK_ERROR("kernel execution failed");

				chain_kernel<type><<<(nc + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel,0,stream_calc>>>(gpu_chain_heads, d_dt, reach_flag, sync_interval, d_offset, d_new_strent, d_new_tau_CD, d_write_time, correlator_type, d_rand_used, d_tau_CD_used_CD, d_tau_CD_used_SD, steps_count % stressarray_count);
				CUT_CHECK_ERROR("kernel execution failed");

				hipUnbindTexture(t_a_QN);
				hipUnbindTexture(t_a_tCD);
			} else { //For even number of steps
				hipBindTextureToArray(t_a_QN, d_b_QN, channelDesc4);
				cudaBindSurfaceToArray(s_b_QN, d_a_QN);
				hipBindTextureToArray(t_a_tCD, d_b_tCD, channelDesc1);
				cudaBindSurfaceToArray(s_b_tCD, d_a_tCD);
				hipBindTextureToArray(t_a_R1, d_b_R1, channelDesc4);
				cudaBindSurfaceToArray(s_b_R1, d_a_R1);
				if (texture_flag == true){
					cudaBindSurfaceToArray(s_corr, d_corr_b);
				} else {
					cudaBindSurfaceToArray(s_corr, d_corr_a);
				}

				strent_kernel<type><<<dimGrid, dimBlock,0,stream_calc>>>(gpu_chain_heads, d_dt, d_offset, d_new_strent, d_new_tau_CD);
				CUT_CHECK_ERROR("kernel execution failed");

				chain_kernel<type><<<(nc + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel,0,stream_calc>>>(gpu_chain_heads, d_dt, reach_flag, sync_interval, d_offset, d_new_strent, d_new_tau_CD, d_write_time, correlator_type, d_rand_used, d_tau_CD_used_CD, d_tau_CD_used_SD, steps_count % stressarray_count);
				CUT_CHECK_ERROR("kernel execution failed");

				hipUnbindTexture(t_a_QN);
				hipUnbindTexture(t_a_tCD);
			}

			steps_count++;

			// update progress bar
			if (steps_count % 50 == 0) {
				hipStreamSynchronize(stream_calc);
				hipMemcpyAsync(tbuffer, d_write_time, sizeof(int) * nc, hipMemcpyDeviceToHost, stream_calc);
				hipStreamSynchronize(stream_calc);
				int sumt = 0;
				for (int i = 0; i < nc; i++)
					sumt += tbuffer[i];
				*progress_bar = (int)(100.0f * sumt / (nc) / reach_time);
				//cout << "\r" << *progress_bar << "%\t ";
			}

			// check for rand refill
			if (steps_count % uniformrandom_count == 0) {
				random_textures_refill(nc, 0);
				steps_count = 0;
			}


			if (steps_count % stressarray_count == 0) {
				hipStreamSynchronize(stream_calc);
				hipStreamSynchronize(stream_update);
				hipUnbindTexture(t_corr);
				if (texture_flag==true){
					hipBindTextureToArray(t_corr, d_corr_b, channelDesc4);
					texture_flag = false;
				} else {
					hipBindTextureToArray(t_corr, d_corr_a, channelDesc4);
					texture_flag = true;
				}
				if (type==0){
					update_correlator<<<(nc + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel,0,stream_update>>>((corr)->gpu_corr, stressarray_count, correlator_type);
				}
				if (type==1){
					flow_stress<<<(nc + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel,0,stream_update>>>((corr)->gpu_corr, stressarray_count, stress_average);
				}
			}

			// check for reached time
			hipStreamSynchronize(stream_calc);
			hipMemcpyAsync(rtbuffer, reach_flag, sizeof(float) * nc, hipMemcpyDeviceToHost, stream_calc);
			hipStreamSynchronize(stream_calc);
			float sumrt = 0;
			for (int i = 0; i < nc; i++)
				sumrt += rtbuffer[i];
			reach_flag_all = (sumrt == nc);

			// stop, if run_flag is changed from outside
			if (*run_flag == false)
				return -1;
		}
	}	//loop ends

	if (type==0){
		if (steps_count % stressarray_count != 0) {
			hipStreamSynchronize(stream_calc);
			hipStreamSynchronize(stream_update);
			hipUnbindTexture(t_corr);
			if (texture_flag==true){
				hipBindTextureToArray(t_corr, d_corr_b, channelDesc4);
				texture_flag = false;
			} else {
				hipBindTextureToArray(t_corr, d_corr_a, channelDesc4);
				texture_flag = true;
			}
			if (type==0){
				update_correlator<<<(nc + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel,0,stream_update>>>((corr)->gpu_corr, steps_count, correlator_type);
			}
			if (type==1){
				flow_stress<<<(nc + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel,0,stream_update>>>((corr)->gpu_corr, steps_count, stress_average);
			}
		}
	}

	block_time = reach_time;
	hipHostUnregister(rtbuffer);
	hipHostFree(rtbuffer);
	deactivate_block();
	hipStreamDestroy(stream_update);
	hipStreamDestroy(stream_calc);

	if (type==1){
		hipDeviceSynchronize();
		for (int i=0; i < nsteps; i++){
			cout << "\n" << stress_average[i].x / nc << "\t" << stress_average[i].y / nc << "\t" << stress_average[i].z / nc;
		}
	}
	return 0;
}

int ensemble_block::equilibrium_calc(double length, int correlator_type, bool* run_flag, int *progress_bar, int np, float* t, float* x){
	transfer_to_device();
	hipMemset(d_write_time, 0, sizeof(int) * nc);
	if(time_step<0>(length, correlator_type, run_flag, progress_bar)==-1) return -1;
	int *tint = new int[np];
	float *x_buf = new float[np];
	corr->calc(tint, x_buf, correlator_type);
	transfer_from_device();
	for (int j = 0; j < corr->npcorr; j++) {
		t[j] = tint[j];
		x[j] += x_buf[j] / N_cha;
	}
	delete[] x_buf;
	delete[] tint;
	return 0;
}

// utility functions
//h means host(cpu) declarations
//host copies of gpu inline access functions
//purpose-- to recreate latest chain conformations from gpu memory (to account for delayed dynamics)
int hmake_offset(int i, int offset) {
	//offset&0xffff00)>>8 offset_index
	//offset&0xff-1; offset_dir
	return i >= ((offset & 0xffff00) >> 8) ? i + ((offset & 0xff) - 1) : i;
}
int hoffset_index(int offset) {
	return ((offset & 0xffff00) >> 8);
}

int hoffset_dir(int offset) {
	return (offset & 0xff) - 1;
}

bool fetch_hnew_strent(int i, int offset) {
	return (i == hoffset_index(offset)) && (hoffset_dir(offset) == -1);
}

void ensemble_block::transfer_to_device(){
	hipDeviceSynchronize();
	hipMemcpy(gpu_chain_heads, chain_heads, sizeof(scalar_chains) * nc, hipMemcpyHostToDevice);

	//blank delayed dynamics arrays
	hipMemset(d_dt, 0, sizeof(float) * nc);
	hipMemset(d_offset, 0xff, sizeof(float) * nc);

}

void ensemble_block::transfer_from_device() { //copy chains back
	hipDeviceSynchronize();

	//NOTE accounts for delayed dynamics

	hipMemcpy(chain_heads, gpu_chain_heads, sizeof(scalar_chains) * nc, hipMemcpyDeviceToHost);

	//delayed dynamics
	int *h_offset = new int[nc];
	float4 *h_new_strent = new float4[nc];
	float *h_new_tau_CD = new float[nc];
	hipMemcpy(h_offset, d_offset, sizeof(int) * nc, hipMemcpyDeviceToHost);
	hipMemcpy(h_new_strent, d_new_strent, sizeof(float4) * nc, hipMemcpyDeviceToHost);
	hipMemcpy(h_new_tau_CD, d_new_tau_CD, sizeof(float) * nc, hipMemcpyDeviceToHost);
	for (int i = 0; i < nc; i++) {
		if (hoffset_dir(h_offset[i]) == -1) {
			for (int j = z_max - 1; j > 0; j--) {
				chains.QN[i * z_max + j] = chains.QN[i * z_max + hmake_offset(j, h_offset[i])];
				chains.tau_CD[i * z_max + j] = chains.tau_CD[i * z_max + hmake_offset(j, h_offset[i])];
			}
			chains.QN[i * z_max + hoffset_index(h_offset[i])] = h_new_strent[i];
			chains.tau_CD[i * z_max + hoffset_index(h_offset[i])] = h_new_tau_CD[i];
		} else {
			for (int j = 0; j < z_max - 2; j++) {
				chains.QN[i * z_max + j] = chains.QN[i * z_max + hmake_offset(j, h_offset[i])];
				chains.tau_CD[i * z_max + j] = chains.tau_CD[i * z_max + hmake_offset(j, h_offset[i])];
			}
		}
	}

	delete[] h_offset;
	delete[] h_new_strent;
	delete[] h_new_tau_CD;
}

stress_plus ensemble_block::calc_stress(int *r_chain_count) {

	hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dn_cha_per_call), &nc, sizeof(int)));
	hipMemcpy2DToArray(d_a_QN, 0, 0, chains.QN, z_max * sizeof(float) * 4, z_max * sizeof(float) * 4, nc, hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();
	hipBindTextureToArray(t_a_QN, d_a_QN, channelDesc4);
	stress_calc<<<(nc + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel>>>(gpu_chain_heads, d_dt, d_offset, d_new_strent, chains.QN, z_max * 4 * nc);
	CUT_CHECK_ERROR("kernel execution failed");
	hipUnbindTexture(t_a_QN);
	hipMemcpy2DFromArray(chains.QN, sizeof(float) * z_max * 4, d_a_QN, 0, 0, z_max * sizeof(float) * 4, nc, hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();

	float4 *stress_buf = new float4[nc * 2];
	hipMemcpyFromArray(stress_buf, d_stress, 0, 0, nc * sizeof(float4) * 2, hipMemcpyDeviceToHost);
	float4 sum_stress = make_float4(0.0f, 0.0f, 0.0f, 0.0f); //stress: xx,yy,zz,xy
	float4 sum_stress2 = make_float4(0.0f, 0.0f, 0.0f, 0.0f); //stress: yz,xz; Lpp, Ree
	scalar_chains* tchain_heads;
	tchain_heads = new scalar_chains[nc];

	hipMemcpy(tchain_heads, gpu_chain_heads, sizeof(scalar_chains) * nc, hipMemcpyDeviceToHost);
	int chain_count = nc;
	for (int j = 0; j < nc; j++) {
		if (tchain_heads[j].stall_flag == 0) {
			if (!isnan(stress_buf[j * 2].x)) {
				sum_stress.x += stress_buf[j * 2].x;
				sum_stress.y += stress_buf[j * 2].y;
				sum_stress.z += stress_buf[j * 2].z;
				sum_stress.w += stress_buf[j * 2].w;
				sum_stress2.x += stress_buf[j * 2 + 1].x;
				sum_stress2.y += stress_buf[j * 2 + 1].y;
				sum_stress2.z += stress_buf[j * 2 + 1].z;
				sum_stress2.w += stress_buf[j * 2 + 1].w;
//				cout<<"stress chain "<<j<<'\t'<<sum_stress.x<<'\t'<<sum_stress.y<<'\t'<<sum_stress.z<<'\t'<<sum_stress.w<<'\n';
			} else {
				chain_count--;
				cout << "chain stall " << j << '\n';  //TODO output gloval index
			}
		} else {
			chain_count--;
			cout << "chain stall " << j << '\n';    //TODO output gloval index
		}
	}
	stress_plus rs;
	rs.xx = sum_stress.x / chain_count;
	rs.yy = sum_stress.y / chain_count;
	rs.zz = sum_stress.z / chain_count;
	rs.xy = sum_stress.w / chain_count;
	rs.yz = sum_stress2.x / chain_count;
	rs.zx = sum_stress2.y / chain_count;
	rs.Lpp = sum_stress2.z / chain_count;
	rs.Z = sum_stress2.w / chain_count;
	delete[] stress_buf;
	delete[] tchain_heads;
	*r_chain_count = chain_count;
	return rs;
}

void ensemble_block::activate_block() {
	//prepares block for performing time evolution
	//i.e. copies chain conformations to working memory
	hipChannelFormatDesc channelDesc1 = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dn_cha_per_call), &nc, sizeof(int)));

	float tf = block_time;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_universal_time), &tf, sizeof(float)));

	if (!(steps_count & 0x00000001)) {
		hipMemcpy2DToArray(d_a_QN, 0, 0, chains.QN, z_max * sizeof(float) * 4, z_max * sizeof(float) * 4, nc, hipMemcpyDeviceToDevice);
		hipMemcpy2DToArray(d_a_tCD, 0, 0, chains.tau_CD, z_max * sizeof(float), z_max * sizeof(float), nc, hipMemcpyDeviceToDevice);
		hipMemcpyToArray(d_a_R1, 0, 0, chains.R1, sizeof(float) * 4 * nc, hipMemcpyDeviceToDevice);
	} else {
		hipMemcpy2DToArray(d_b_QN, 0, 0, chains.QN, z_max * sizeof(float) * 4, z_max * sizeof(float) * 4, nc, hipMemcpyDeviceToDevice);
		hipMemcpy2DToArray(d_b_tCD, 0, 0, chains.tau_CD, z_max * sizeof(float), z_max * sizeof(float), nc, hipMemcpyDeviceToDevice);
		hipMemcpyToArray(d_b_R1, 0, 0, chains.R1, sizeof(float) * 4 * nc, hipMemcpyDeviceToDevice);
	}
	hipDeviceSynchronize();
}

void ensemble_block::deactivate_block() {
	//copies chain conformations to storing memory

	if (!(steps_count & 0x00000001)) {
		hipMemcpy2DFromArray(chains.QN, sizeof(float) * z_max * 4, d_a_QN, 0, 0, z_max * sizeof(float) * 4, nc, hipMemcpyDeviceToDevice);
		hipMemcpy2DFromArray(chains.tau_CD, sizeof(float) * z_max, d_a_tCD, 0, 0, z_max * sizeof(float), nc, hipMemcpyDeviceToDevice);
		hipMemcpyFromArray(chains.R1, d_a_R1, 0, 0, sizeof(float) * 4 * nc, hipMemcpyDeviceToDevice);
	} else {
		hipMemcpy2DFromArray(chains.QN, sizeof(float) * z_max * 4, d_b_QN, 0, 0, z_max * sizeof(float) * 4, nc, hipMemcpyDeviceToDevice);
		hipMemcpy2DFromArray(chains.tau_CD, sizeof(float) * z_max, d_b_tCD, 0, 0, z_max * sizeof(float), nc, hipMemcpyDeviceToDevice);
		hipMemcpyFromArray(chains.R1, d_b_R1, 0, 0, sizeof(float) * 4 * nc, hipMemcpyDeviceToDevice);
	}
	hipDeviceSynchronize();
}

ensemble_block::~ensemble_block() {    //free memory
	hipFree(chains.QN);
	hipFree(chains.tau_CD);
	hipFree(chains.R1);

	delete[] chain_heads;

	hipFree(gpu_chain_heads);

	hipFree(d_dt);
	hipFree(reach_flag);
	hipFree(d_offset);
	hipFree(d_new_strent);
	hipFree(d_new_tau_CD);

	hipFree(d_write_time);

	if (corr != NULL) {
		delete corr;
	}
}
