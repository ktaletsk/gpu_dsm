#include "hip/hip_runtime.h"
// Copyright 2015 Marat Andreev, Konstantin Taletskiy, Maria Katzarova
// 
// This file is part of gpu_dsm.
// 
// gpu_dsm is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// at your option) any later version.
// 
// gpu_dsm is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with gpu_dsm.  If not, see <http://www.gnu.org/licenses/>.
#include "textures_surfaces.h"
#include "chain.h"
#include "gpu_random.h"
#include "ensemble_kernel.cu"
#include "ensemble_block.h"
#include "correlator.h"
#include <vector>
#include "timer.h"
#define max_sync_interval 1 //Doi-Takimoto requires synchronising every timestep

//variable arrays, that are common for all the blocks
gpu_Ran *d_random_gens; // device random number generators
gpu_Ran *d_random_gens2; //first is used to pick jump process, second is used for creation of new entanglements
//temporary arrays fpr random numbers sequences
hipArray* d_uniformrand; // uniform random number supply //used to pick jump process
hipArray* d_taucd_gauss_rand_CD; // 1x uniform + 3x normal distributed random number supply// used for creating entanglements by SD
hipArray* d_taucd_gauss_rand_SD; // used for creating entanglements by SD
int steps_count = 0;    //time step count
int *d_tau_CD_used_SD;
int *d_tau_CD_used_CD;
int *d_rand_used;
int *d_value_found;
int* d_shift_found;
float* d_add_rand;
int* d_end_list;		// binary list of creations/destructions at the end of each arm in ensemble
int* d_end_counter;		// counter for d_end_list
int* d_destroy_list;	// list of paired chains for destroyed entanglements at the end of each arm in ensemble
int* d_destroy_counter;
int* d_destroy_list_2;	// list of paired chains for destroyed entanglements at the end of each arm in ensemble
int* d_destroy_counter_2;
int* d_create_counter;
int* d_doi_weights;

// these arrays used by time evolution kernels
hipArray* d_a_QN; //device arrays for vector part of chain conformations
hipArray* d_a_tCD;
hipArray* d_a_tcr;
hipArray* d_a_pair_chains;
hipArray* d_b_QN;
hipArray* d_b_tCD;
hipArray* d_b_tcr;
hipArray* d_b_pair_chains;
hipArray* d_a_R1;
hipArray* d_b_R1;
hipArray* d_corr_a;
hipArray* d_corr_b;
hipArray* d_ft;
hipArray* d_arm_index;

hipArray* d_sum_W; // sum of probabilities for each entanglement
hipArray* d_sum_W_sorted;
hipArray* d_stress; // stress calculation temp array

// There are two arrays with the vector part of the chain conformations  on device.
// And there is only one array with the scalar part of the chain conformations
// Every timestep the vector part is copied from one array to another.
// The coping is done in entanglement parallel portion of the code
// This allows to use textures/surfaces and speeds up memory access
// The scalar part(chain headers) is updated in the chain parallel portion of the code
// Chain headers occupy less memory,and there are no specific memory access technics for them.

//random shuffle
template< class RandomIt >
void random_s(RandomIt first, RandomIt last, Ran* eran)
{
	typename std::iterator_traits<RandomIt>::difference_type i, n;
	n = last - first;
	for (i = n - 1; i > 0; --i) {
		using std::swap;
		swap(first[i], first[(int)(eran->flt()* RAND_MAX) % (i + 1)]);
	}
}

void ensemble_block::init(int nc_, vector_chains chains_, scalar_chains* chain_heads_, int nsteps_){
	//allocates arrays, copies chain conformations to device
	//ensemble_call_block *cb pointer for call block structure, just ref parameter
	//int nc  is a number of chains in this ensemble_call_block.
	//sstrentp chains, chain_head* chain_heads pointers to array with the chain conformations

	nc = nc_;
	nsteps = nsteps_;
	chains = chains_;
	chain_heads = chain_heads_;
	block_time = universal_time;

	//blank dynamics probabalities
	float *buffer = new float[4 * z_max * nc];
	memset(buffer, 0, sizeof(float4) * z_max * nc);
	hipMemcpy2DToArray(d_sum_W, 0, 0, buffer, z_max * sizeof(int), z_max * sizeof(int), nc, hipMemcpyHostToDevice);
	delete[] buffer;

	float *buffer3 = new float[z_max * nc];
	memset(buffer3, 0, sizeof(float) * z_max * nc);
	hipMemcpy2DToArray(d_sum_W_sorted, 0, 0, buffer3, z_max * sizeof(float), z_max * sizeof(float), nc, hipMemcpyHostToDevice);
	delete[] buffer3;

	float *buffer2 = new float[nc];
	memset(buffer2, 0, sizeof(float) * nc);
	hipMemcpyToArray(d_ft, 0, 0, buffer2, nc * sizeof(float), hipMemcpyHostToDevice);
	delete[] buffer2;

	// allocating device arrays
	hipMalloc(&d_dt, sizeof(float) * nc);
	hipMemset(d_dt, 0, sizeof(float) * nc);

	hipMalloc(&d_offset, sizeof(int) * nc * narms);
	hipMemset(d_offset, 0xff, sizeof(float) * nc * narms);

	hipMalloc(&reach_flag, sizeof(float) * nc);
	hipMalloc(&d_new_strent, sizeof(float) * 4 * nc);
	hipMalloc(&d_new_tau_CD, sizeof(float) * nc);
	hipMalloc(&d_new_cr_time, sizeof(float) * nc);
	hipMalloc(&d_new_pair, sizeof(float) * nc);

	//hipMemset(d_new_pair, -1.0f, sizeof(float) * nc);

	int s = ceil(log((float)nsteps/correlator_size)/log(correlator_res)) + 1; //number of correlator levels
	cout << "number of correlator levels" << '\t' << s << '\n' << '\n';
	corr = new correlator(nc, s);//Allocate memory for c_correlator structure in cb
	hipMalloc((void**) &d_write_time, sizeof(int) * nc);//Allocated memory on device for correlator time for every chain in block
	hipMemset(d_write_time, 0, sizeof(int) * nc);	//Initialize d_correlator_time with zeros

//	hipMallocManaged((void**)&(stress_average), sizeof(float4) * nsteps * nc); //4vectors for every tau_k for every chain
	CUT_CHECK_ERROR("kernel execution failed");
}

template<int type> int  ensemble_block::time_step(double reach_time, int correlator_type, bool* run_flag, int *progress_bar) {
	//bind textures/surfaces, perform time evolution, unbind textures/surfaces

	//Declare and create streams for parallel correlator update
	hipStream_t stream_calc1;
	hipStream_t stream_calc2;
	hipStream_t stream_calc3;
	hipStream_t stream_calc4;
	hipStream_t stream_update;
	hipStreamCreate(&stream_calc1);
	hipStreamCreate(&stream_calc2);
	hipStreamCreate(&stream_calc3);
	hipStreamCreate(&stream_calc4);
	hipStreamCreate(&stream_update);

	hipChannelFormatDesc channelDesc1 = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat); //to read float4
	hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat); //to read float

	//loop preparing
	dim3 dimBlock(tpb_strent_kernel/4, tpb_strent_kernel);
	dim3 dimGrid((z_max + dimBlock.x - 1) / dimBlock.x, (nc + dimBlock.y - 1) / dimBlock.y);

	dim3 dimBlockFlat(z_max, 1);
	dim3 dimGridFlat((z_max + dimBlockFlat.x - 1) / dimBlockFlat.x, nc);

	steps_count = 0;
	activate_block();

	float time_step_interval = reach_time - block_time;
	int number_of_syncs = int(floor((time_step_interval - 0.5) / max_sync_interval)) + 1;

	float *rtbuffer;
	hipHostMalloc(&rtbuffer, sizeof(float)*nc);

	int *tbuffer;
	hipHostMalloc(&tbuffer, sizeof(int)*nc);

	float *entbuffer;
	hipHostMalloc(&entbuffer, sizeof(float)*nc);

	std::vector<unsigned long long> enttime_bins (20000, 0);

	int Narms_ensemble = nc*narms;

	bool texture_flag = true;

	cudaBindSurfaceToArray(s_arm_index, d_arm_index);

	//random number generator for pairing chains
	Ran eran_2(2);
	//Initialize random
	eran_2.seed(narms * N_cha);

	//Loop begins
	for (int i_sync = 0; i_sync < number_of_syncs; i_sync++) {
		
		float sync_interval = max_sync_interval;
		if ((i_sync + 1) == number_of_syncs)
			sync_interval = time_step_interval - i_sync * max_sync_interval;

		bool reach_flag_all = false;
		hipMemset(reach_flag, 0, sizeof(float) * nc);

		//update universal_time on device
		float tf = block_time + i_sync * max_sync_interval;

		//printf("\nStep %i, time %f", i_sync, tf);
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_universal_time), &tf, sizeof(float)));
		
		while (!reach_flag_all) {
			//cout << "\nSteps_count " << steps_count << " a/b " << !(steps_count & 0x00000001);
			if (!(steps_count & 0x00000001)) { //For odd number of steps

				hipBindTextureToArray(t_a_QN, d_a_QN, channelDesc4);
				cudaBindSurfaceToArray(s_b_QN, d_b_QN);
				hipBindTextureToArray(t_a_tCD, d_a_tCD, channelDesc1);
				cudaBindSurfaceToArray(s_b_tCD, d_b_tCD);
				hipBindTextureToArray(t_a_tcr, d_a_tcr, channelDesc1);
				cudaBindSurfaceToArray(s_b_tcr, d_b_tcr);
				hipBindTextureToArray(t_a_pair, d_a_pair_chains, channelDesc1);
				cudaBindSurfaceToArray(s_b_pair, d_b_pair_chains);
				hipBindTextureToArray(t_a_R1, d_a_R1, channelDesc4);
				cudaBindSurfaceToArray(s_b_R1, d_b_R1);
			}
			else { //For even number of steps
				hipBindTextureToArray(t_a_QN, d_b_QN, channelDesc4);
				cudaBindSurfaceToArray(s_b_QN, d_a_QN);
				hipBindTextureToArray(t_a_tCD, d_b_tCD, channelDesc1);
				cudaBindSurfaceToArray(s_b_tCD, d_a_tCD);
				hipBindTextureToArray(t_a_tcr, d_b_tcr, channelDesc1);
				cudaBindSurfaceToArray(s_b_tcr, d_a_tcr);
				hipBindTextureToArray(t_a_pair, d_b_pair_chains, channelDesc1);
				cudaBindSurfaceToArray(s_b_pair, d_a_pair_chains);
				hipBindTextureToArray(t_a_R1, d_b_R1, channelDesc4);
				cudaBindSurfaceToArray(s_b_R1, d_a_R1);
			}

			if (texture_flag == true) {
				cudaBindSurfaceToArray(s_corr, d_corr_b);
			}
			else {
				cudaBindSurfaceToArray(s_corr, d_corr_a);
			}

			strent_kernel<type> <<<dimGrid, dimBlock, 0, stream_calc1>>> (chain_heads, d_dt, d_offset, d_new_strent, d_new_tau_CD, d_new_cr_time, d_new_pair);
			CUT_CHECK_ERROR("kernel execution failed");
			boundary2_kernel<3> <<<(Narms_ensemble + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel, 0, stream_calc2 >>> (chain_heads, d_offset, d_new_strent, d_new_tau_CD);
			CUT_CHECK_ERROR("kernel execution failed");
			boundary1_kernel <<<(Narms_ensemble + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel, 0, stream_calc3 >>> (chain_heads, d_offset, d_new_strent);
			CUT_CHECK_ERROR("kernel execution failed");
			hipStreamSynchronize(stream_calc2);
			hipStreamSynchronize(stream_calc3);

			chain_control_kernel<type> <<<(nc + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel, 0, stream_calc4 >>> (chain_heads, d_dt, reach_flag, sync_interval, d_offset, d_new_strent, d_write_time, correlator_type, steps_count % stressarray_count);
			CUT_CHECK_ERROR("kernel execution failed");

			scan_kernel <<<dimGridFlat, dimBlockFlat, 2 * z_max * sizeof(int), stream_calc1 >>> (chain_heads, d_rand_used, d_value_found, d_shift_found, d_add_rand);
			CUT_CHECK_ERROR("kernel execution failed");
			
			hipMemcpyAsync(rtbuffer, reach_flag, sizeof(float) * nc, hipMemcpyDeviceToHost, stream_calc4);
			hipStreamSynchronize(stream_calc4);

			chain_kernel<type> <<<(nc + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel, 0, stream_calc1 >>> (chain_heads, d_dt, reach_flag, sync_interval, d_offset, d_new_strent, d_new_tau_CD, d_new_cr_time, d_write_time, correlator_type, d_rand_used, d_tau_CD_used_CD, d_tau_CD_used_SD, d_value_found, d_shift_found, d_add_rand, d_end_list, d_end_counter, d_destroy_list, d_destroy_counter, d_create_counter, d_new_pair);
			CUT_CHECK_ERROR("kernel execution failed");

			float sumrt = 0;
			for (int i = 0; i < nc; i++)
				sumrt += rtbuffer[i];
			reach_flag_all = (sumrt == nc);

			hipUnbindTexture(t_a_QN);
			hipUnbindTexture(t_a_tCD);
			hipUnbindTexture(t_a_tcr);
			hipUnbindTexture(t_a_pair);
			hipUnbindTexture(t_a_R1);

			steps_count++;

//			copy entanglement lifetimes
			cudaMemcpyFromArrayAsync(entbuffer, d_ft, 0, 0, sizeof(float) * nc, hipMemcpyDeviceToHost, stream_calc1);
			hipStreamSynchronize(stream_calc1);
			for (int i = 0; i < nc; i++){
				if ((entbuffer[i]>0.0) && (entbuffer[i]<20.0)){
					enttime_bins[floor(entbuffer[i]*1000)]++;
				}
			}

			//Should be shared between blocks...

			// update progress bar
//			if (steps_count % 50 == 0) {
//				hipStreamSynchronize(stream_calc);
//				hipMemcpyAsync(tbuffer, d_write_time, sizeof(int) * nc, hipMemcpyDeviceToHost, stream_calc);
//				hipStreamSynchronize(stream_calc);
//				int sumt = 0;
//				for (int i = 0; i < nc; i++)
//					sumt += tbuffer[i];
//				*progress_bar = (int)(100.0f * sumt / (nc) / reach_time);
//				//cout << "\r" << *progress_bar << "%\t ";
//			}

			// check for reached time
//			hipMemcpyAsync(rtbuffer, reach_flag, sizeof(float) * nc, hipMemcpyDeviceToHost, stream_calc1);

			// check for rand refill
			if (steps_count % uniformrandom_count == 0) {
				random_textures_refill(nc, 0);
				steps_count = 0;
			}

			if (steps_count % stressarray_count == 0) {
				hipStreamSynchronize(stream_calc1);
				hipUnbindTexture(t_corr);
				if (texture_flag == true) {
					hipBindTextureToArray(t_corr, d_corr_b, channelDesc4);
					texture_flag = false;
				}
				else {
					hipBindTextureToArray(t_corr, d_corr_a, channelDesc4);
					texture_flag = true;
				}
				if (type == 0 && correlator_type == 0) {
					update_correlator <<<(nc + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel, 0, stream_update >>>((corr)->gpu_corr, stressarray_count, correlator_type);
				}
				if (correlator_type == 1 || correlator_type == 2) {
					flow_stress <<<(nc + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel, 0, stream_update >>>((corr)->gpu_corr, stressarray_count, stress_average, nc);
				}
			}

			// stop, if run_flag is changed from outside
			if (*run_flag == false)
				return -1;
		}
		//print (for dynamic pairing)
		hipStreamSynchronize(stream_calc1);
		
		hipMemset(d_end_list, 0, sizeof(int) * nc * narms);
		hipMemset(d_end_counter, 0, sizeof(int) * nc * narms);

		hipMemset(d_destroy_list_2, 0, sizeof(int) * nc * 10);
		hipMemset(d_destroy_counter_2, 0, sizeof(int) * nc);

		for (int i = 0; i < nc; i++) {
			for (int arm = 0; arm < narms; arm++) {
				for (int k = 0; k < d_destroy_counter[narms * i + arm]; k++) {
					int ch = d_destroy_list[10 * (narms*i + arm) + k];
					d_destroy_list_2[10 * ch + d_destroy_counter_2[ch]] = i;
					d_destroy_counter_2[ch]++;
				}
			}
		}
		
		int n_destroy_iterations = 0;
		for (int i = 0; i < nc; i++) {
			if (d_destroy_counter_2[i] > n_destroy_iterations)
				n_destroy_iterations = d_destroy_counter_2[i];
		}
		n_destroy_iterations = n_destroy_iterations + n_destroy_iterations % 2;

		hipMemset(d_destroy_list, 0, sizeof(int) * nc * narms * 10);
		hipMemset(d_destroy_counter, 0, sizeof(int) * nc * narms);

		std::vector<std::pair<int, int> > NewPairs;
		//Number of new entanglements created by every arm in the ensemble
		//cout << "Number of new entanglements created by every arm in the ensemble\n";
		for (int i = 0; i < nc; i++) {
			int create_counter = 0;
			for (int arm = 0; arm < narms; arm++)
				create_counter += d_create_counter[i*narms + arm];
				//cout << "Chain " << i << "\tCounter " << create_counter << "\n";
			for (int c = 0; c < create_counter; c++) {
				NewPairs.push_back(std::make_pair(i, -1));
			}
		}
		hipMemset(d_create_counter, 0, sizeof(int) * nc * narms);

		random_s(NewPairs.begin(), NewPairs.end(), &eran_2);

		//cout << "Now making " << n_destroy_iterations << " destroy iterations\n";
		// Destroy second parts of entanglement pairs
		int add_steps_count = 0;
		while (add_steps_count < n_destroy_iterations) {
			//cout << "\nIteration " << add_steps_count << "a/b " << !((steps_count + add_steps_count) & 0x00000001) << "\n";
			if (!((steps_count + add_steps_count) & 0x00000001)) { //For odd number of steps

				hipBindTextureToArray(t_a_QN, d_a_QN, channelDesc4);
				cudaBindSurfaceToArray(s_b_QN, d_b_QN);
				hipBindTextureToArray(t_a_tCD, d_a_tCD, channelDesc1);
				cudaBindSurfaceToArray(s_b_tCD, d_b_tCD);
				hipBindTextureToArray(t_a_tcr, d_a_tcr, channelDesc1);
				cudaBindSurfaceToArray(s_b_tcr, d_b_tcr);
				hipBindTextureToArray(t_a_pair, d_a_pair_chains, channelDesc1);
				cudaBindSurfaceToArray(s_b_pair, d_b_pair_chains);
				hipBindTextureToArray(t_a_R1, d_a_R1, channelDesc4);
				cudaBindSurfaceToArray(s_b_R1, d_b_R1);
			}
			else { //For even number of steps
				hipBindTextureToArray(t_a_QN, d_b_QN, channelDesc4);
				cudaBindSurfaceToArray(s_b_QN, d_a_QN);
				hipBindTextureToArray(t_a_tCD, d_b_tCD, channelDesc1);
				cudaBindSurfaceToArray(s_b_tCD, d_a_tCD);
				hipBindTextureToArray(t_a_tcr, d_b_tcr, channelDesc1);
				cudaBindSurfaceToArray(s_b_tcr, d_a_tcr);
				hipBindTextureToArray(t_a_pair, d_b_pair_chains, channelDesc1);
				cudaBindSurfaceToArray(s_b_pair, d_a_pair_chains);
				hipBindTextureToArray(t_a_R1, d_b_R1, channelDesc4);
				cudaBindSurfaceToArray(s_b_R1, d_a_R1);
			}

			hipMemset(d_value_found, -1, sizeof(int) * nc);
			strent_doi_sync_kernel<type> <<<dimGrid, dimBlock, 0, stream_calc1 >>> (chain_heads, d_dt, d_offset, d_new_strent, d_new_tau_CD, d_new_cr_time, d_new_pair, d_destroy_list_2, d_destroy_counter_2, d_value_found, add_steps_count);
			CUT_CHECK_ERROR("kernel execution failed");
			chain_doi_destroy_kernel<type> <<<(nc + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel, 0, stream_calc1 >>> (chain_heads, d_dt, sync_interval, d_offset, d_new_strent, d_new_tau_CD, d_new_cr_time, d_value_found, d_end_list, d_end_counter, d_destroy_list_2, d_destroy_counter_2);
			CUT_CHECK_ERROR("kernel execution failed");
			
			hipUnbindTexture(t_a_QN);
			hipUnbindTexture(t_a_tCD);
			hipUnbindTexture(t_a_tcr);
			hipUnbindTexture(t_a_pair);
			hipUnbindTexture(t_a_R1);
			add_steps_count++;
		}
		
		hipStreamSynchronize(stream_calc1);
		// Search for pairs for new entanglements
		add_steps_count = 0;

		//initialize weights
		chain_doi_initial_weights <<<(nc + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel, 0, stream_calc1 >>> (chain_heads, d_doi_weights);
		CUT_CHECK_ERROR("kernel execution failed");

		//cout << "\na/b " << !((steps_count + add_steps_count) & 0x00000001);
		if (!((steps_count + add_steps_count) & 0x00000001)) { //For odd number of steps

			hipBindTextureToArray(t_a_QN, d_a_QN, channelDesc4);
			cudaBindSurfaceToArray(s_b_QN, d_b_QN);
			hipBindTextureToArray(t_a_tCD, d_a_tCD, channelDesc1);
			cudaBindSurfaceToArray(s_b_tCD, d_b_tCD);
			hipBindTextureToArray(t_a_tcr, d_a_tcr, channelDesc1);
			cudaBindSurfaceToArray(s_b_tcr, d_b_tcr);
			hipBindTextureToArray(t_a_pair, d_a_pair_chains, channelDesc1);
			cudaBindSurfaceToArray(s_b_pair, d_b_pair_chains);
			hipBindTextureToArray(t_a_R1, d_a_R1, channelDesc4);
			cudaBindSurfaceToArray(s_b_R1, d_b_R1);
		}
		else { //For even number of steps
			hipBindTextureToArray(t_a_QN, d_b_QN, channelDesc4);
			cudaBindSurfaceToArray(s_b_QN, d_a_QN);
			hipBindTextureToArray(t_a_tCD, d_b_tCD, channelDesc1);
			cudaBindSurfaceToArray(s_b_tCD, d_a_tCD);
			hipBindTextureToArray(t_a_tcr, d_b_tcr, channelDesc1);
			cudaBindSurfaceToArray(s_b_tcr, d_a_tcr);
			hipBindTextureToArray(t_a_pair, d_b_pair_chains, channelDesc1);
			cudaBindSurfaceToArray(s_b_pair, d_a_pair_chains);
			hipBindTextureToArray(t_a_R1, d_b_R1, channelDesc4);
			cudaBindSurfaceToArray(s_b_R1, d_a_R1);
		}

		//restrict double pairing of chains
		strent_doi_sync_2_kernel<type> <<<dimGrid, dimBlock, 0, stream_calc1 >>> (chain_heads, d_dt, d_offset, d_new_strent, d_new_tau_CD, d_new_cr_time, d_new_pair, d_doi_weights);
		CUT_CHECK_ERROR("kernel execution failed");

		hipStreamSynchronize(stream_calc1);

		//cout << "\nCreating new pairs";

		hipMemset(d_destroy_list_2, 0, sizeof(int) * nc * 10);
		hipMemset(d_destroy_counter_2, 0, sizeof(int) * nc);

		for (unsigned pair = 0; pair < NewPairs.size(); pair++) {
			d_destroy_counter_2[NewPairs[pair].first]++;
		}

		int n_create_iterations = 0;
		for (int i = 0; i < nc; i++) {
			if (d_destroy_counter_2[i] > n_create_iterations)
				n_create_iterations = d_destroy_counter_2[i];
			//cout << "\nCreate " << d_destroy_counter_2[i] << " pairs with chain " << i;
		}
		//cout << "\nNeed iterations: " << n_create_iterations;

		hipStreamSynchronize(stream_calc1);
		//scan_weights_kernel
		for (int counter = 0; counter < n_create_iterations; counter++) {
			chain_doi_scan_weights << <(nc + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel, 0, stream_calc1 >> > (chain_heads, d_rand_used, d_doi_weights, d_destroy_list_2, d_destroy_counter_2, counter);
		}
		hipStreamSynchronize(stream_calc1);
	
		//update weights
		//for (int i = 0; i < nc; i++) {
		//	if (d_doi_weights[nc*new_dynamic_pair + i] != 0)
		//		d_doi_weights[nc*new_dynamic_pair + i]--;
		//}
		//d_doi_weights[nc*NewPairs[pair].first + new_dynamic_pair] = 0;
		//d_doi_weights[nc*new_dynamic_pair + NewPairs[pair].first] = 0;

		//apply found pairs to the first part of pair
		chain_doi_label_pairs_kernel<type> <<<(nc + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel, 0, stream_calc1 >>> (chain_heads, d_dt, sync_interval, d_offset, d_new_strent, d_new_tau_CD, d_new_cr_time, d_new_pair, d_rand_used, d_value_found, d_end_list, d_end_counter, d_destroy_list_2, d_destroy_counter_2, d_doi_weights);
		CUT_CHECK_ERROR("kernel execution failed");

		hipUnbindTexture(t_a_QN);
		hipUnbindTexture(t_a_tCD);
		hipUnbindTexture(t_a_tcr);
		hipUnbindTexture(t_a_pair);
		hipUnbindTexture(t_a_R1);

		//hipDeviceSynchronize();
		hipStreamSynchronize(stream_calc1);
		//creeate list of pairs to link
		//cout << "\nCreate new entanglements to complete the pair on the following chains:\n";
		hipMemset(d_destroy_list_2, 0, sizeof(int) * nc * 10);
		hipMemset(d_destroy_counter_2, 0, sizeof(int) * nc);

		for (unsigned pair = 0; pair < NewPairs.size(); pair++) {
			d_destroy_list_2[10 * NewPairs[pair].second + d_destroy_counter_2[NewPairs[pair].second]]=NewPairs[pair].first;
			d_destroy_counter_2[NewPairs[pair].second]++;
		}

		//int n_create_iterations = 0;
		//for (int i = 0; i < nc; i++) {
		//	//cout << "\nCreate pair on chain " << i << " with ";
		//	//for (int c = 0; c < d_destroy_counter_2[i]; c++) {
		//	//	cout << d_destroy_list_2[10 * i + c] << " ";
		//	//}
		//	//cout << "\n";
		//	if (d_destroy_counter_2[i] > n_create_iterations)
		//		n_create_iterations = d_destroy_counter_2[i];
		//}
		n_create_iterations = n_create_iterations + 3 + (n_create_iterations + 1) % 2;
		add_steps_count++;

		//hipDeviceSynchronize();
		hipStreamSynchronize(stream_calc1);
		//start creating second halves of pairs
		//cout << "\nNow making " << n_create_iterations << " create iterations\n";
		while (add_steps_count < n_create_iterations) {
			//cout << "\nIteration " << add_steps_count << " a/b " << !((steps_count + add_steps_count) & 0x00000001) << "\n";
			if (!((steps_count + add_steps_count) & 0x00000001)) { //For odd number of steps

				hipBindTextureToArray(t_a_QN, d_a_QN, channelDesc4);
				cudaBindSurfaceToArray(s_b_QN, d_b_QN);
				hipBindTextureToArray(t_a_tcr, d_a_tcr, channelDesc1);
				cudaBindSurfaceToArray(s_b_tcr, d_b_tcr);
				hipBindTextureToArray(t_a_pair, d_a_pair_chains, channelDesc1);
				cudaBindSurfaceToArray(s_b_pair, d_b_pair_chains);
			}
			else { //For even number of steps
				hipBindTextureToArray(t_a_QN, d_b_QN, channelDesc4);
				cudaBindSurfaceToArray(s_b_QN, d_a_QN);
				hipBindTextureToArray(t_a_tcr, d_b_tcr, channelDesc1);
				cudaBindSurfaceToArray(s_b_tcr, d_a_tcr);
				hipBindTextureToArray(t_a_pair, d_b_pair_chains, channelDesc1);
				cudaBindSurfaceToArray(s_b_pair, d_a_pair_chains);
			}
			strent_doi_sync_3_kernel<type> <<< dimGrid, dimBlock, 0, stream_calc1 >>> (chain_heads, d_dt, d_offset, d_new_strent, d_new_tau_CD, d_new_cr_time, d_new_pair);
			CUT_CHECK_ERROR("kernel execution failed");
			chain_doi_create_kernel<type> <<<(nc + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel, 0, stream_calc1 >>> (chain_heads, d_dt, sync_interval, d_offset, d_new_strent, d_new_tau_CD, d_new_cr_time, d_rand_used, d_tau_CD_used_CD, d_destroy_list_2, d_destroy_counter_2, add_steps_count-1, d_new_pair);
			CUT_CHECK_ERROR("kernel execution failed");
			//hipDeviceSynchronize();
			hipStreamSynchronize(stream_calc1);
			hipUnbindTexture(t_a_QN);
			hipUnbindTexture(t_a_tCD);
			hipUnbindTexture(t_a_tcr);
			hipUnbindTexture(t_a_pair);
			hipUnbindTexture(t_a_R1);

			add_steps_count++;
		}

	}	//loop ends


	if (type==0){
		if (steps_count % stressarray_count != 0) {
			hipStreamSynchronize(stream_calc1);
			hipStreamSynchronize(stream_update);
			hipUnbindTexture(t_corr);
			if (texture_flag==true){
				hipBindTextureToArray(t_corr, d_corr_b, channelDesc4);
				texture_flag = false;
			} else {
				hipBindTextureToArray(t_corr, d_corr_a, channelDesc4);
				texture_flag = true;
			}
			if (type==0 && correlator_type==0){
				update_correlator<<<(nc + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel,0,stream_update>>>((corr)->gpu_corr, steps_count, correlator_type);
			}
			if (correlator_type==1 || correlator_type==2){
				flow_stress<<<(nc + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel,0,stream_update>>>((corr)->gpu_corr, steps_count, stress_average, nc);
			}
		}
	}
	block_time = reach_time;
	hipHostUnregister(rtbuffer);
	hipHostFree(rtbuffer);
	deactivate_block();
	hipStreamDestroy(stream_update);
	hipStreamDestroy(stream_calc1);

	//output entanglement lifetime distribution
	unsigned long long enttime_sum = 0;
	for (int it = 0; it < enttime_bins.size(); ++it) {
		enttime_sum += enttime_bins[it];
	}

	ofstream lifetime_file;
	lifetime_file.open(filename_ID("fdt", false));
	//unsigned long long enttime_run_sum = 0;
	for (int it = 0; it < enttime_bins.size(); ++it){
		if (enttime_bins[it] != 0){
			//enttime_run_sum += enttime_bins[it];
			//lifetime_file << powf(10.0f,it/1000.0f-10.0f) << '\t' << 1.0f - (float)enttime_run_sum / (float)enttime_sum << '\n';
			lifetime_file << it << '\t' << enttime_bins[it] << '\n';
		}
	}
	lifetime_file.close();
	return 0;
}

int ensemble_block::equilibrium_calc(double length, int correlator_type, bool* run_flag, int *progress_bar, int np, float* t, float* x){
	transfer_to_device();
	hipMemset(d_write_time, 0, sizeof(int) * nc);
	if(time_step<0>(length, correlator_type, run_flag, progress_bar)==-1) return -1;
	int *tint = new int[np];
	float *x_buf = new float[np];
	corr->calc(tint, x_buf, correlator_type);
	transfer_from_device();
	for (int j = 0; j < corr->npcorr; j++) {
		t[j] = tint[j];
		x[j] += x_buf[j] / N_cha;
	}
	delete[] x_buf;
	delete[] tint;
	return 0;
}

// utility functions
//h means host(cpu) declarations
//host copies of gpu inline access functions
//purpose-- to recreate latest chain conformations from gpu memory (to account for delayed dynamics)
int hmake_offset(int i, int offset) {
	//offset&0xffff00)>>8 offset_index
	//offset&0xff-1; offset_dir
	return i >= ((offset & 0xffff00) >> 8) ? i + ((offset & 0xff) - 1) : i;
}
int hoffset_index(int offset) {
	return ((offset & 0xffff00) >> 8);
}

int hoffset_dir(int offset) {
	return (offset & 0xff) - 1;
}

bool fetch_hnew_strent(int i, int offset) {
	return (i == hoffset_index(offset)) && (hoffset_dir(offset) == -1);
}

void ensemble_block::transfer_to_device(){
	hipDeviceSynchronize();

	//blank delayed dynamics arrays
	hipMemset(d_dt, 0, sizeof(float) * nc);
	hipMemset(d_offset, 0xff, sizeof(float) * nc * narms);

}

void ensemble_block::transfer_from_device() { //copy chains back
	hipDeviceSynchronize();

	//NOTE accounts for delayed dynamics

	//delayed dynamics
	int *h_offset = new int[nc*narms];
	float4 *h_new_strent = new float4[nc];
	float *h_new_tau_CD = new float[nc];
	hipMemcpy(h_offset, d_offset, sizeof(int) * nc * narms, hipMemcpyDeviceToHost);
	hipMemcpy(h_new_strent, d_new_strent, sizeof(float4) * nc, hipMemcpyDeviceToHost);
	hipMemcpy(h_new_tau_CD, d_new_tau_CD, sizeof(float) * nc, hipMemcpyDeviceToHost);
	int run_sum;
	for (int i = 0; i < nc; i++) {
		run_sum = 0;
		for (int arm=0; arm<narms; arm++){
			if (hoffset_dir(h_offset[i*narms+arm]) == -1) {
				for (int j = NK_arms[arm] - 1; j > 0; j--) {
					chains.QN[i * z_max + run_sum + j] = chains.QN[i * z_max + hmake_offset(j + run_sum, h_offset[i*narms+arm])];
					chains.tau_CD[i * z_max + run_sum + j] = chains.tau_CD[i * z_max + hmake_offset(j + run_sum, h_offset[i*narms+arm])];
				}
				chains.QN[i * z_max + run_sum + hoffset_index(h_offset[i*narms+arm])] = h_new_strent[i];
				chains.tau_CD[i * z_max + run_sum + hoffset_index(h_offset[i*narms+arm])] = h_new_tau_CD[i];
			} else {
				for (int j = 0; j < NK_arms[arm] - 2; j++) {
					chains.QN[i * z_max + run_sum + j] = chains.QN[i * z_max + hmake_offset(j + run_sum, h_offset[i*narms+arm])];
					chains.tau_CD[i * z_max + run_sum + j] = chains.tau_CD[i * z_max + hmake_offset(j + run_sum, h_offset[i*narms+arm])];
				}
			}
			run_sum += NK_arms[arm];
		}
	}

	delete[] h_offset;
	delete[] h_new_strent;
	delete[] h_new_tau_CD;
}

stress_plus ensemble_block::calc_stress(int *r_chain_count) {

	hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dn_cha_per_call), &nc, sizeof(int)));
	hipMemcpy2DToArray(d_a_QN, 0, 0, chains.QN, z_max * sizeof(float) * 4, z_max * sizeof(float) * 4, nc, hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();
	hipBindTextureToArray(t_a_QN, d_a_QN, channelDesc4);
	stress_calc<<<(nc + tpb_chain_kernel - 1) / tpb_chain_kernel, tpb_chain_kernel>>>(chain_heads, d_dt, d_offset, d_new_strent, chains.QN, z_max * 4 * nc);
	CUT_CHECK_ERROR("kernel execution failed");
	hipUnbindTexture(t_a_QN);
	hipMemcpy2DFromArray(chains.QN, sizeof(float) * z_max * 4, d_a_QN, 0, 0, z_max * sizeof(float) * 4, nc, hipMemcpyDeviceToDevice);
	hipDeviceSynchronize();

	float4 *stress_buf = new float4[nc * 2];
	hipMemcpyFromArray(stress_buf, d_stress, 0, 0, nc * sizeof(float4) * 2, hipMemcpyDeviceToHost);
	float4 sum_stress = make_float4(0.0f, 0.0f, 0.0f, 0.0f); //stress: xx,yy,zz,xy
	float4 sum_stress2 = make_float4(0.0f, 0.0f, 0.0f, 0.0f); //stress: yz,xz; Lpp, Ree
	scalar_chains* tchain_heads;
	tchain_heads = new scalar_chains[nc];

	hipMemcpy(tchain_heads, chain_heads, sizeof(scalar_chains) * nc, hipMemcpyDeviceToHost);
	int chain_count = nc;
	for (int j = 0; j < nc; j++) {
		if (tchain_heads[j].stall_flag == 0) {
			if (!isnan(stress_buf[j * 2].x)) {
				sum_stress.x += stress_buf[j * 2].x;
				sum_stress.y += stress_buf[j * 2].y;
				sum_stress.z += stress_buf[j * 2].z;
				sum_stress.w += stress_buf[j * 2].w;
				sum_stress2.x += stress_buf[j * 2 + 1].x;
				sum_stress2.y += stress_buf[j * 2 + 1].y;
				sum_stress2.z += stress_buf[j * 2 + 1].z;
				sum_stress2.w += stress_buf[j * 2 + 1].w;
//				cout<<"stress chain "<<j<<'\t'<<sum_stress.x<<'\t'<<sum_stress.y<<'\t'<<sum_stress.z<<'\t'<<sum_stress.w<<'\n';
			} else {
				chain_count--;
				cout << "chain stall " << j << '\n';  //TODO output gloval index
			}
		} else {
			chain_count--;
			cout << "chain stall " << j << '\n';    //TODO output gloval index
		}
	}
	stress_plus rs;
	rs.xx = sum_stress.x / chain_count;
	rs.yy = sum_stress.y / chain_count;
	rs.zz = sum_stress.z / chain_count;
	rs.xy = sum_stress.w / chain_count;
	rs.yz = sum_stress2.x / chain_count;
	rs.zx = sum_stress2.y / chain_count;
	rs.Lpp = sum_stress2.z / chain_count;
	rs.Z = sum_stress2.w / chain_count;
	delete[] stress_buf;
	delete[] tchain_heads;
	*r_chain_count = chain_count;
	return rs;
}

void ensemble_block::activate_block() {
	//prepares block for performing time evolution
	//i.e. copies chain conformations to working memory
	hipChannelFormatDesc channelDesc1 = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dn_cha_per_call), &nc, sizeof(int)));

	float tf = block_time;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_universal_time), &tf, sizeof(float)));

	if (!(steps_count & 0x00000001)) {
		hipMemcpy2DToArray(d_a_QN, 0, 0, chains.QN, z_max * sizeof(float) * 4, z_max * sizeof(float) * 4, nc, hipMemcpyDeviceToDevice);
		hipMemcpy2DToArray(d_a_tCD, 0, 0, chains.tau_CD, z_max * sizeof(float), z_max * sizeof(float), nc, hipMemcpyDeviceToDevice);
		hipMemcpy2DToArray(d_a_tcr, 0, 0, chains.tau_cr, z_max * sizeof(float), z_max * sizeof(float), nc, hipMemcpyDeviceToDevice);
		hipMemcpy2DToArray(d_a_pair_chains, 0, 0, chains.pair_chain, z_max * sizeof(int), z_max * sizeof(int), nc, hipMemcpyDeviceToDevice);
		hipMemcpyToArray(d_a_R1, 0, 0, chains.R1, sizeof(float) * 4 * nc, hipMemcpyDeviceToDevice);
	} else {
		hipMemcpy2DToArray(d_b_QN, 0, 0, chains.QN, z_max * sizeof(float) * 4, z_max * sizeof(float) * 4, nc, hipMemcpyDeviceToDevice);
		hipMemcpy2DToArray(d_b_tCD, 0, 0, chains.tau_CD, z_max * sizeof(float), z_max * sizeof(float), nc, hipMemcpyDeviceToDevice);
		hipMemcpy2DToArray(d_b_tcr, 0, 0, chains.tau_cr, z_max * sizeof(float), z_max * sizeof(float), nc, hipMemcpyDeviceToDevice);
		hipMemcpy2DToArray(d_b_pair_chains, 0, 0, chains.pair_chain, z_max * sizeof(int), z_max * sizeof(int), nc, hipMemcpyDeviceToDevice);
		hipMemcpyToArray(d_b_R1, 0, 0, chains.R1, sizeof(float) * 4 * nc, hipMemcpyDeviceToDevice);
	}
	hipDeviceSynchronize();
}

void ensemble_block::deactivate_block() {
	//copies chain conformations to storing memory

	if (!(steps_count & 0x00000001)) {
		hipMemcpy2DFromArray(chains.QN, sizeof(float) * z_max * 4, d_a_QN, 0, 0, z_max * sizeof(float) * 4, nc, hipMemcpyDeviceToDevice);
		hipMemcpy2DFromArray(chains.tau_CD, sizeof(float) * z_max, d_a_tCD, 0, 0, z_max * sizeof(float), nc, hipMemcpyDeviceToDevice);
		hipMemcpy2DFromArray(chains.tau_cr, sizeof(float) * z_max, d_a_tcr, 0, 0, z_max * sizeof(float), nc, hipMemcpyDeviceToDevice);
		hipMemcpy2DFromArray(chains.pair_chain, sizeof(int) * z_max, d_a_pair_chains, 0, 0, z_max * sizeof(int), nc, hipMemcpyDeviceToDevice);
		hipMemcpyFromArray(chains.R1, d_a_R1, 0, 0, sizeof(float) * 4 * nc, hipMemcpyDeviceToDevice);
	} else {
		hipMemcpy2DFromArray(chains.QN, sizeof(float) * z_max * 4, d_b_QN, 0, 0, z_max * sizeof(float) * 4, nc, hipMemcpyDeviceToDevice);
		hipMemcpy2DFromArray(chains.tau_CD, sizeof(float) * z_max, d_b_tCD, 0, 0, z_max * sizeof(float), nc, hipMemcpyDeviceToDevice);
		hipMemcpy2DFromArray(chains.tau_cr, sizeof(float) * z_max, d_b_tcr, 0, 0, z_max * sizeof(float), nc, hipMemcpyDeviceToDevice);
		hipMemcpy2DFromArray(chains.pair_chain, sizeof(int) * z_max, d_b_pair_chains, 0, 0, z_max * sizeof(int), nc, hipMemcpyDeviceToDevice);
		hipMemcpyFromArray(chains.R1, d_b_R1, 0, 0, sizeof(float) * 4 * nc, hipMemcpyDeviceToDevice);
	}
	hipDeviceSynchronize();
}

ensemble_block::~ensemble_block() {    //free memory
	hipFree(d_dt);
	hipFree(reach_flag);
	hipFree(d_offset);
	hipFree(d_new_strent);
	hipFree(d_new_tau_CD);
	hipFree(d_new_cr_time);
	hipFree(d_new_pair);

	hipFree(d_write_time);

	if (corr != NULL) {
		delete corr;
	}
}
