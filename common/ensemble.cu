#include "hip/hip_runtime.h"
// Copyright 2015 Marat Andreev, Konstantin Taletskiy, Maria Katzarova
// 
// This file is part of gpu_dsm.
// 
// gpu_dsm is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// at your option) any later version.
// 
// gpu_dsm is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with gpu_dsm.  If not, see <http://www.gnu.org/licenses/>.

#include "gpu_random.h"
#include <iostream>
#include <stdlib.h>
#include <fstream>
#include <vector>
#include <algorithm>
#include "ensemble.h"
#include "cudautil.h"
#include "cuda_call.h"
#include "textures_surfaces.h"
#include "math.h"
extern char * filename_ID(string filename, bool temp);

extern float mp,Mk;
extern float step;
extern float GEX_table[200000];
extern bool PD_flag;

void random_textures_refill(int n_cha);
void random_textures_fill(int n_cha);

#include "ensemble_call_block.cu"

using namespace std;

extern hipArray* d_gamma_table;
extern hipArray* d_gamma_table_d;

#define chains_per_call 32000
//MAX surface size is 32768

sstrentp chains; // host chain conformations

// and only one array with scalar part chain conformation
// every time the vector part is copied from one array to another
// coping is done in entanglement parallel portion of the code
// this allows to use textures/surfaces, which speeds up memory access
// scalar part(chain headers) are update in the chain parallel portion of the code
// chain headers are occupied much smaller memory, no specific memory access technic are used for them.
// depending one odd or even number of time step were performed,
//one of the get_chains_from_device_# should be used

chain_head* chain_heads; // host device chain headers arrays, store scalar variables of chain conformations

int chain_blocks_number;
ensemble_call_block *chain_blocks;

//host constants
double universal_time;//since chain_head do not store universal time due to SP issues
		      	  	  //see chain.h chain_head for explanation
int N_cha;
int NK;
int z_max;
float Be;
float kxx, kxy, kxz, kyx, kyy, kyz, kzx, kzy, kzz;
bool PD_flag=0;

bool dbug = false;	//true;

//navigation
sstrentp chain_index(const int i) { //absolute navigation i - is a global index of chains i:[0..N_cha-1]
	sstrentp ptr;
	ptr.QN = &(chains.QN[z_max * i]);
	ptr.tau_CD = &(chains.tau_CD[z_max * i]);
	return ptr;
}

sstrentp chain_index(const int bi, const int i) {    //block navigation
	//bi is a block index bi :[0..chain_blocks_number]
	//i - is a chain index in the block bi  i:[0..chains_per_call-1]
	sstrentp ptr;
	ptr.QN = &(chains.QN[z_max * (bi * chains_per_call + i)]);
	ptr.tau_CD = &(chains.tau_CD[z_max * (bi * chains_per_call)]);
	return ptr;
}

void chains_malloc() {
	//setup z_max modification maybe needed for large \beta
	z_max = NK;    // current realization limits z to 2^23
	chain_heads = new chain_head[N_cha];
	chains.QN = new float4[N_cha * z_max];
	chains.tau_CD = new float[N_cha * z_max];
}

void host_chains_init(Ran* eran) {
	chains_malloc();
	cout << "generating chain conformations on host..";
	universal_time=0.0;
	for (int i = 0; i < N_cha; i++) {
		sstrentp ptr = chain_index(i);
		chain_init(&(chain_heads[i]), ptr, NK, z_max, PD_flag, eran);
	}
	cout << "done\n";
}

//preparation of constants/arrays/etc
void gpu_init(int seed, p_cd* pcd, int s) {
	cout << "preparing GPU chain conformations..\n";

	//Copy host constants from host to device
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dBe), &Be, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dnk), &NK, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_z_max), &z_max, sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_kappa_xx), &kxx, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_kappa_xy), &kxy, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_kappa_xz), &kxz, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_kappa_yx), &kyx, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_kappa_yy), &kyy, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_kappa_yz), &kyz, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_kappa_zx), &kzx, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_kappa_zy), &kzy, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_kappa_zz), &kzz, sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_CD_flag), &CD_flag, sizeof(int)));

	float cdtemp;
	if(PD_flag){
		//calculate probability prefactor for polydisperse simulations
		double tem = 0.0f;
		for (int i=0; i+1<1/step; i++){
			p_cd* t_pcd = new p_cd(Be, GEX_table[i]*mp/Mk, NULL);
			tem += (t_pcd->W_CD_destroy_aver());
			delete[] t_pcd;
		}
		cdtemp = step * tem / Be;
	}
	else
		cdtemp = pcd->W_CD_destroy_aver() / Be;

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_CD_create_prefact), &cdtemp, sizeof(float)));
	cout << " device constants done\n";

	int rsz = chains_per_call;
	if (N_cha < chains_per_call)
		rsz = N_cha;
	hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	hipChannelFormatDesc channelDesc1 = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	hipMallocArray(&d_a_QN, &channelDesc4, z_max, rsz, hipArraySurfaceLoadStore);
	hipMallocArray(&d_a_tCD, &channelDesc1, z_max, rsz, hipArraySurfaceLoadStore);

	hipMallocArray(&d_b_QN, &channelDesc4, z_max, rsz, hipArraySurfaceLoadStore);
	hipMallocArray(&d_b_tCD, &channelDesc1, z_max, rsz, hipArraySurfaceLoadStore);

	hipMallocArray(&d_sum_W, &channelDesc1, z_max, rsz, hipArraySurfaceLoadStore);
	hipMallocArray(&d_stress, &channelDesc4, rsz * 2, 0, hipArraySurfaceLoadStore);
	cudaBindSurfaceToArray(s_stress, d_stress);
	cudaBindSurfaceToArray(s_sum_W, d_sum_W);

	cout << "\n";
	cout << " GPU random generator init: \n";
	cout << "  device random generators 1 seeding..";
	hipMalloc(&d_random_gens, sizeof(gpu_Ran) * rsz);

	gr_array_seed(d_random_gens, rsz, seed * rsz); //
	cout << ".done\n";
	cout << "  device random generators 2 seeding..";
	hipMalloc(&d_random_gens2, sizeof(gpu_Ran) * rsz);
	gr_array_seed(d_random_gens2, rsz, (seed + 1) * rsz);
	cout << ".done\n";

	cout << "  preparing random number sequence..";
	random_textures_fill(rsz);
	cout << ".done\n";
	cout << " GPU random generator init done.\n";
	cout << "\n";

	//Calculate number of necessary blocks of chains
	chain_blocks_number = (N_cha + chains_per_call - 1) / chains_per_call;
	cout << " Number of ensemble blocks " << chain_blocks_number << '\n';

	//chain_blocks - array of blocks
	chain_blocks = new ensemble_call_block[chain_blocks_number];
	for (int i = 0; i < chain_blocks_number - 1; i++) {
		init_call_block(&(chain_blocks[i]), chains_per_call, chain_index(i, 0), &(chain_heads[i * chains_per_call]),s);
		cout << "  copying chains to device block " << i + 1 << ". chains in the ensemble block " << chains_per_call << '\n';
	}
	init_call_block(&(chain_blocks[chain_blocks_number - 1]), (N_cha - 1) % chains_per_call + 1, chain_index(chain_blocks_number - 1, 0), &(chain_heads[(chain_blocks_number - 1) * chains_per_call]),s);
	cout << "  copying chains to device block " << chain_blocks_number
			<< ". chains in the ensemble block "
			<< (N_cha - 1) % chains_per_call + 1 << '\n';
	cout << " device chains done\n";

	cout << "init done\n";
}

stress_plus calc_stress() {
	stress_plus tmps = make_stress_plus(0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0);
	int total_chains = 0;
	for (int i = 0; i < chain_blocks_number; i++) {
		int cc;
		stress_plus tmp = calc_stress_call_block(&(chain_blocks[i]), &cc);
		total_chains += cc;
		double w = double(cc);
		tmps = tmps + tmp * w;
	}
	return tmps / total_chains;
}

int gpu_time_step(double reach_time, bool* run_flag) {
	for (int i = 0; i < chain_blocks_number; i++) {
		if(time_step_call_block(reach_time, &(chain_blocks[i]), run_flag)==-1) return -1;
	}
	universal_time=reach_time;
	return 0;
}

void get_chains_from_device()    //Copies chains back to host memory
{
	for (int i = 0; i < chain_blocks_number; i++) {
		get_chain_from_device_call_block(&(chain_blocks[i]));
	}
}

void random_textures_fill(int n_cha) {
	hipChannelFormatDesc channelDesc1 = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);

	hipMallocArray(&(d_taucd_gauss_rand_CD), &channelDesc4, uniformrandom_count, n_cha, hipArraySurfaceLoadStore);
	hipMallocArray(&(d_taucd_gauss_rand_SD), &channelDesc4, uniformrandom_count, n_cha, hipArraySurfaceLoadStore);
	hipMallocArray(&(d_uniformrand), &channelDesc1, uniformrandom_count, n_cha, hipArraySurfaceLoadStore);

	hipMalloc((void**) &d_rand_used, sizeof(int) * n_cha);
	hipMemset(d_rand_used, 0, sizeof(int) * n_cha);
	hipMalloc((void**) &d_tau_CD_used_CD, sizeof(int) * n_cha);
	hipMalloc((void**) &d_tau_CD_used_SD, sizeof(int) * n_cha);
	hipMemset(d_tau_CD_used_CD, 0, sizeof(int) * n_cha);
	hipMemset(d_tau_CD_used_SD, 0, sizeof(int) * n_cha);

	gr_fill_surface_uniformrand(d_random_gens, n_cha, uniformrandom_count, d_uniformrand);
	hipDeviceSynchronize();

	int taucd_gauss_count = uniformrandom_count;
	gr_fill_surface_taucd_gauss_rand(d_random_gens2, n_cha, taucd_gauss_count, false, d_taucd_gauss_rand_CD); //Set array with random numbers
	gr_fill_surface_taucd_gauss_rand(d_random_gens2, n_cha, taucd_gauss_count, true,  d_taucd_gauss_rand_SD);

	hipBindTextureToArray(t_uniformrand, d_uniformrand, channelDesc1);
	hipBindTextureToArray(t_taucd_gauss_rand_CD, d_taucd_gauss_rand_CD, channelDesc4);
	hipBindTextureToArray(t_taucd_gauss_rand_SD, d_taucd_gauss_rand_SD, channelDesc4);
}

void random_textures_refill(int n_cha) {
	if (chain_blocks_number != 1)
		n_cha = chains_per_call;

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipChannelFormatDesc channelDesc4 = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);

	hipUnbindTexture(t_uniformrand);
	gr_fill_surface_uniformrand(d_random_gens, n_cha, uniformrandom_count, d_uniformrand);
	hipMemset(d_rand_used, 0, sizeof(int) * n_cha);
	hipBindTextureToArray(t_uniformrand, d_uniformrand, channelDesc);
	hipDeviceSynchronize();

	//tau_cd gauss 3d vector
	hipUnbindTexture(t_taucd_gauss_rand_CD);
	gr_refill_surface_taucd_gauss_rand(d_random_gens2, n_cha, d_tau_CD_used_CD,false, d_taucd_gauss_rand_CD);
	gr_refill_surface_taucd_gauss_rand(d_random_gens2, n_cha, d_tau_CD_used_SD, true, d_taucd_gauss_rand_SD);
	hipMemset(d_tau_CD_used_CD, 0, sizeof(int) * n_cha);
	hipMemset(d_tau_CD_used_SD, 0, sizeof(int) * n_cha);
	hipBindTextureToArray(t_taucd_gauss_rand_CD, d_taucd_gauss_rand_CD, channelDesc4);
	hipBindTextureToArray(t_taucd_gauss_rand_SD, d_taucd_gauss_rand_SD, channelDesc4);
	hipDeviceSynchronize();
}


int gpu_Gt_PCS(int res, double length, float *&t, float *&x, int s, bool* run_flag, int *progress_bar) {
	//Start simulation
	//Calculate stress with timestep specified
	//Update correlators on the fly for each chain (on GPU)
	//At the end copy results from each chain and average them

	*progress_bar = 1;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_correlator_res), &(res), sizeof(int))); //Copy timestep for calculation

	int np = correlator_size + (s-1) * (correlator_size - (float)correlator_size/(float)correlator_res);

	t = new float[np];
	x = new float[np];

	for (int j = 0; j < np; j++) {
		t[j] = 0.0f;
		x[j] = 0.0f;
	}

	for (int i = 0; i < chain_blocks_number; i++){
		int *tint = new int[np];
		float *x_buf = new float[np];

		get_chain_to_device_call_block(&(chain_blocks[i]));
		hipMemset(chain_blocks[i].d_correlator_time, 0,sizeof(int) * chain_blocks[i].nc);
		if(EQ_time_step_call_block(length, &(chain_blocks[i]),run_flag, progress_bar)==-1) return -1;
		chain_blocks[i].corr->calc(tint, x_buf);
		get_chain_from_device_call_block(&(chain_blocks[i]));

		for (int j = 0; j < chain_blocks[i].corr->npcorr; j++) {
			t[j] = tint[j];
			x[j] += x_buf[j] / N_cha;
		}
		delete[] x_buf;
		delete[] tint;
	}

	ofstream G_file;
	G_file.open(filename_ID("G",false));
	cout << "\n";
	int actual_np = (np - correlator_size + floor(length/((float)res*pow((float)correlator_res,(s-1)))));
	for (int j = 0; j < actual_np; j++) {
		cout << t[j] << '\t' << x[j] << '\n';
		G_file << t[j] << '\t' << x[j] << '\n';
	}
	G_file.close();
	delete[] t;
	delete[] x;
	return 0;
}


void save_to_file(char *filename) {
	ofstream file(filename, ios::out | ios::binary);
//	ofstream file(filename, ios::out);
	if (file.is_open()) {
		file.write((char*) &N_cha, sizeof(int));

		for (int i = 0; i < N_cha; i++) {
			save_to_file(file, chain_index(i), chain_heads[i]);
		}

	} else
		cout << "file error\n";
}

void save_Z_distribution_to_file(string filename /*char *filename*/, bool cumulative) {

	ofstream file(filename.c_str(), ios::out);
	if (file.is_open()) {
//		file<<"Number of chains: "<<N_cha<<"\n";

		//Calculation of Z distribution in ensemble

		//Search for maximum and minimum of Z
		int Zmin = chain_heads[0].Z;
		int Zmax = chain_heads[0].Z;
		for (int i = 0; i < N_cha; i++) {
			if (chain_heads[i].Z > Zmax)
				Zmax = chain_heads[i].Z;
			if (chain_heads[i].Z < Zmin)
				Zmin = chain_heads[i].Z;
		}

		//Sum up coinciding Z
		float P[N_cha];
		for (int j = Zmin; j <= Zmax; j++) {
			P[j]=0.0f;
			int counter=0;
			for (int i = 0; i < N_cha; i++) {
				if (chain_heads[i].Z == j && !cumulative)
					counter++;
				if (chain_heads[i].Z <= j && cumulative)
					counter++;
			}
			P[j] = (float) counter / N_cha;
		}

		for (int i = Zmin; i <= Zmax; i++)
			file << i << "\t" << P[i] << "\n";
	} else
		cout << "file error\n";
}

void save_N_distribution_to_file(string filename, bool cumulative) {
	ofstream file(filename.c_str(), ios::out);
	if (file.is_open()) {
		//Search for maximum and minimum of N across all strands in all chains
		int Nmin = chain_index(0).QN[0].w;
		int Nmax = chain_index(0).QN[0].w;
		int Nstr = 0;
		for (int i = 0; i < N_cha; i++) {
			for (int j = 0; j < chain_heads[i].Z; j++) {
				if (chain_index(i).QN[j].w > Nmax)
					Nmax = chain_index(i).QN[j].w;
				if (chain_index(i).QN[j].w < Nmin)
					Nmin = chain_index(i).QN[j].w;
				if (chain_index(i).QN[j].w == 0)
					cout << "Zero length strand in chain " << i << ", #" << j << "\n";
			}
			Nstr += chain_heads[i].Z;
		}

		//Sum up coinciding N
		float P[Nstr];
		for (int n = Nmin; n <= Nmax; n++) {
			P[n]=0.0f;
			int counter=0;
			for (int i = 0; i < N_cha; i++) {
				for (int j = 0; j < chain_heads[i].Z; j++) {
					if (chain_index(i).QN[j].w == n && !cumulative)
						counter++;
					if (chain_index(i).QN[j].w <= n && cumulative)
						counter++;
				}
			}
			P[n] += (float)counter / Nstr;
			file << n << "\t" << P[n] << "\n";
		}
	} else
		cout << "file error\n";
}

int compare(const void * a, const void * b) {
	float fa = *(const float*) a;
	float fb = *(const float*) b;
	return (fa > fb) - (fa < fb);
}

void save_Q_distribution_to_file(string filename, bool cumulative) {
	ofstream file(filename.c_str(), ios::out);
	if (file.is_open()) {
		//Search for maximal and minimal value of |Q| across all strands in all chains

		int Nstr = 0;
		int prev[N_cha];

		for (int i = 0; i < N_cha; i++) {
			Nstr += chain_heads[i].Z - 2;
			if (i == 0)
				prev[i] = 0;
			else
				prev[i] = prev[i - 1] + chain_heads[i - 1].Z - 2;
		}

		//Calculating strand vector lengths
		float Q[Nstr];
		for (int i = 0; i < N_cha; i++) {
			for (int j = 1; j < chain_heads[i].Z - 1; j++) {
				Q[prev[i] + j - 1] = sqrt(chain_index(i).QN[j].x * chain_index(i).QN[j].x + chain_index(i).QN[j].y * chain_index(i).QN[j].y + chain_index(i).QN[j].z * chain_index(i).QN[j].z);
				if (Q[prev[i] + j - 1] < 0)
					cout << "NaN detected at strand " << prev[i] + j << "\n";
			}
		}
		cout << "\n";

		//Sort vector lenghts
		qsort(Q, Nstr, sizeof(float), compare);

		//Calculate probabilities
		float P[Nstr];
		for (int i = 0; i < Nstr; i++) {
			if (i != 0) {
				if (Q[i] == Q[i - 1])
					P[i - 1] = (float) i / (float) Nstr;
			}
			P[i] = (float) i / (float) Nstr;
		}
		int quant = 10;
		for (int i = 0; i < Nstr / quant; i++) {
			file << Q[i * quant] << "\t" << P[i * quant] << "\n";
		}
	} else
		cout << "file error\n";
}

void load_from_file(char *filename) {

	chains_malloc();
	ifstream file(filename, ios::in | ios::binary);
	if (file.is_open()) {
		int ti;
		file.read((char*) &ti, sizeof(int));
		if (ti != N_cha) {
			cout << "ensemble size mismatch\n";
			exit(2);
		}

		for (int i = 0; i < N_cha; i++) {
			load_from_file(file, chain_index(i), &chain_heads[i]);
		}

	} else
		cout << "file error\n";
}

void gpu_clean() {
	cout << "Memory cleanup.. ";

	delete[] chain_blocks;
	delete[] chain_heads;
	delete[] (chains.QN);
	delete[] (chains.tau_CD);

	hipFreeArray(d_a_QN);
	hipFreeArray(d_a_tCD);
	hipFreeArray(d_b_QN);
	hipFreeArray(d_b_tCD);
	hipFreeArray(d_sum_W);
	hipFreeArray(d_stress);

	hipFreeArray(d_uniformrand);
	hipFreeArray(d_taucd_gauss_rand_CD);
	hipFreeArray(d_taucd_gauss_rand_SD);

	hipFree(d_tau_CD_used_SD);
	hipFree(d_tau_CD_used_CD);
	hipFree(d_rand_used);
	hipFree(d_random_gens);
	hipFree(d_random_gens2);

	hipFree(d_gamma_table);
	hipFree(d_gamma_table_d);
	hipDeviceReset();
	cout << "done.\n";
}
