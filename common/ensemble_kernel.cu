#include "hip/hip_runtime.h"
// Copyright 2015 Marat Andreev, Konstantin Taletskiy, Maria Katzarova
// 
// This file is part of gpu_dsm.
// 
// gpu_dsm is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// at your option) any later version.
// 
// gpu_dsm is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with gpu_dsm.  If not, see <http://www.gnu.org/licenses/>.

// Short intro
//
// CUDA devices possess enormous computation capabilities,
// however memory access (especially writing) is relatively slow.
// Unfortunately DSM flow simulation require to update significant part
// of chain conformations variables every time step, which normally bottlenecks the performance.
// First time conformation updated when flow deformation of strand orientation vectors is applied, second time when jump process is applied.
// If the jump process is SD shift, only two neighboring N_i must be updated,
// but in case entanglement creation/destruction major portion of chain conformation
// arrays must be moved. On GPU it is a very expensive operation,
// almost as expensive as updating {Q_i} during deformation.
// Thus we combined two conformation updates into one.
// It is done through "delayed dynamics". This means that jump process is not applied
// immediately, but information about it stored  in temporary variables until deformation applied. 
// Next time step shifting of arrays applied simultaneously together with flow deformation.

#ifndef _ENSEMBLE_KERNEL_
#define _ENSEMBLE_KERNEL_

#if defined(_MSC_VER)
#define uint unsigned int
#endif

#define tpb_chain_kernel 256
#define tpb_strent_kernel 32

#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n) ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
#endif

#include "textures_surfaces.h"
#include "chain.h"

//d means device variables
__constant__ float d_universal_time;
__constant__ float dBe;
__constant__ int dnk;
__constant__ int dnk_arms[100];
__constant__ int d_z_max; //limits actual array size. might come useful for large beta values and for polydisperse systems
__constant__ int d_z_max_arms[100];//limits for each arm
__constant__ int d_indeces_arms[100];
__constant__ int d_narms;
__constant__ int dn_cha_per_call; //number of chains in this call. cannot be bigger than chains_per_call
__constant__ float d_kappa_xx, d_kappa_xy, d_kappa_xz, d_kappa_yx, d_kappa_yy,d_kappa_yz, d_kappa_zx, d_kappa_zy, d_kappa_zz;

//CD constants
__constant__ int d_CD_flag;
__constant__ float d_CD_create_prefact;
__constant__ int d_correlator_res;

#define PRINT_DEBUG false

#define PROBS_CUTOFF 10000.0f
// delayed dynamics --- how does it work:
// There are entanglement parallel portion of the code and chain parallel portion.
// The entanglement parallel part applies flow deformation and calculates jump process probabilities.
// The chain parallel part picks one of the jump processes, generates a new orientation vector and a tau_CD if needed.
// It applies only some simpliest chain conformation changes(SD shifting).
// The Information about complex chain conformation changes(entanglement creation/destruction) is stored in temp arrays d_offset, d_new_strent,d_new_tau_CD.
// Complex changes are applied next time step by entanglement parallel part.

//float4 math operators
inline __device__ void operator+=(float4 &a, float4 b) {
	a.x += b.x;
	a.y += b.y;
	a.z += b.z;
	a.w += b.w;
}
inline __host__ __device__ float4 operator/(float4 a, float b) {
	return make_float4(a.x / b, a.y / b, a.z / b, a.w / b);
}
inline __host__ __device__ float4 operator*(float4 a, float4 b) {
	return make_float4(a.x * b.x, a.y * b.y, a.z * b.z, a.w * b.w);
}
inline __host__ __device__ float4 operator*(float4 a, float b) {
	return make_float4(a.x * b, a.y * b, a.z * b, a.w * b);
}
inline __host__ __device__ float4 operator+(float4 a, float4 b) {
	return make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}
inline __host__ __device__ float4 operator-(float4 a, float4 b) {
	return make_float4(a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w);
}

//offset in 2 component vector {shifting starting index, shifting direction}
//offset stores both components in the one int variable
//index in first 3 bytes, direction in last byte
__device__ __forceinline__ int offset_code(int offset_index, int offset_dir) {
	return (offset_dir + 1) | (offset_index << 8);
}

// returns i or i+/- 1 from offset
__device__ __forceinline__ int make_offset(int i, int offset) {
	//offset&0xffff00)>>8 offset_index
	//offset&0xff-1; offset_dir
	return i >= ((offset & 0xffff00) >> 8) ? i + ((offset & 0xff) - 1) : i;
}

//returns components of offset
__device__ __forceinline__ int offset_index(int offset) {
	return ((offset & 0xffff00) >> 8);
}

__device__ __forceinline__ int offset_dir(int offset) {
	return (offset & 0xff) - 1;
}

//returns true if d_new_strent should be inserted at index i
__device__ __forceinline__ bool fetch_new_strent(int i, int offset) {
	return (i == offset_index(offset)) && (offset_dir(offset) == -1);
}

//deformation
__device__   __forceinline__ float4 kappa(const float4 QN, const float dt) {//Qx is different for consitency with old version
	return make_float4(
			QN.x + dt * d_kappa_xx * QN.x + dt * d_kappa_xy * QN.y + dt * d_kappa_xz * QN.z,
			QN.y + dt * d_kappa_yx * QN.x + dt * d_kappa_yy * QN.y + dt * d_kappa_yz * QN.z,
			QN.z + dt * d_kappa_zx * QN.x + dt * d_kappa_zy * QN.y + dt * d_kappa_zz * QN.z, QN.w);
}

//The entanglement parallel part of the code
//2D kernel: i- entanglement index j - chain index
template<int type> __global__ __launch_bounds__(tpb_strent_kernel*tpb_strent_kernel) void strent_kernel(scalar_chains* chain_heads, float *tdt, int *d_offset, float4 *d_new_strent, float *d_new_tau_CD, float* d_new_cr_time)
{
	//Calculate kernel index
	int i = blockIdx.x * blockDim.x + threadIdx.x;//strent index
	int j = blockIdx.y * blockDim.y + threadIdx.y;//chain index

	//Check if kernel index is outside boundaries
	if ((j >= dn_cha_per_call) || (i >= d_z_max))
		return;
	int arm=0;
	int run_sum=0;
	for (int k=0; i>=run_sum; k++){
		run_sum+=d_z_max_arms[k];
		arm = k;
	}
	int ii = i-run_sum+d_z_max_arms[arm];
	int jj = j*d_narms+arm;

	int tz = chain_heads[j].Z[arm]; //Current chain size
	surf2Dwrite((int)(ii<tz), s_arm_index, 4 * i, j);

	if (ii >= tz) //Check if entaglement index is over chain size
		return;

	//When new entaglements are created we need to shift index +1(destruction, skip one strent), 0(nothing happens) or -1(new strent created before)
	int oft = d_offset[jj]; //Offset for current chain

	float4 QN = fetch_new_strent(i, oft) ? d_new_strent[j] : tex2D(t_a_QN, make_offset(i, oft), j);

	float tcd=0;
	if (d_CD_flag)	tcd = fetch_new_strent(i, oft) ? d_new_tau_CD[j] : tex2D(t_a_tCD, make_offset(i, oft), j);

	float t_cr = fetch_new_strent(i, oft) ? d_new_cr_time[j] : tex2D(t_a_tcr, make_offset(i, oft), j);

	float dt;
	if (type==1){//transform
		dt = tdt[j];
		QN = kappa(QN, dt);
	}

	//printf("\ni=%i\tarm=%i\tstrent=%i\tQ=%f\t%f\t%f\tN=%f\tT=%f", i, arm, ii, QN.x, QN.y, QN.z, QN.w, d_universal_time + chain_heads[j].time);

	//fetch next strent
	if ((ii > 0) && (ii < tz - 1)) {
		int4 wsh = make_int4(0, 0, 0, 0);
		float4 QN2 = fetch_new_strent(i + 1, oft) ? d_new_strent[j] : tex2D(t_a_QN, make_offset(i + 1, oft), j); //Q for next strent

		if (type==1){//transform
			QN2 = kappa(QN2, dt);
		}
		//w_shift probability calc

		float Q = QN.x * QN.x + QN.y * QN.y + QN.z * QN.z;
		float Q2 = QN2.x * QN2.x + QN2.y * QN2.y + QN2.z * QN2.z;

		if (QN2.w > 1.0f) { //N=1 mean that shift is not possible, also ot will lead to dividing on zero error
			float sig1 = __fdividef(0.75f, QN.w * (QN.w + 1)); //fdivedf - fast divide float
			float sig2 = __fdividef(0.75f, QN2.w * (QN2.w - 1));
			float prefact1 = (Q == 0.0f) ? 1.0f : __fdividef(QN.w, (QN.w + 1));
			float prefact2 = (Q2 == 0.0f) ? 1.0f : __fdividef(QN2.w, (QN2.w - 1));
			float f1 = (ii == 0) ? 2.0f * QN.w + 0.5f : QN.w;
			float f2 = (ii == tz-2) ? 2.0f * QN2.w - 0.5f : QN2.w;
			float friction = __fdividef(2.0f, f1 + f2);
			wsh.x = (int)(PROBS_CUTOFF*friction * __powf(prefact1 * prefact2, 0.75f)* __expf(Q * sig1 - Q2 * sig2));
		}
		if (QN.w > 1.0f) {//N=1 mean that shift is not possible, also ot will lead to dividing on zero error
			float sig1 = __fdividef(0.75f, QN.w * (QN.w - 1.0f));
			float sig2 = __fdividef(0.75f, QN2.w * (QN2.w + 1.0f));
			float prefact1 = (Q == 0.0f) ? 1.0f : __fdividef(QN.w, (QN.w - 1.0f));
			float prefact2 = (Q2 == 0.0f) ? 1.0f : __fdividef(QN2.w, (QN2.w + 1.0f));
			float f1 = (ii == 0) ? 2.0f * QN.w - 0.5f : QN.w;
			float f2 = (ii == tz-2) ? 2.0f * QN2.w + 0.5f : QN2.w;
			float friction = __fdividef(2.0f, f1 + f2);
			wsh.y = (int)(PROBS_CUTOFF*friction * __powf(prefact1 * prefact2, 0.75f) * __expf(-Q * sig1 + Q2 * sig2));
		}
		if (d_CD_flag)	wsh.z = (int)(PROBS_CUTOFF*tcd);
		if (d_CD_flag)	wsh.w = (int)(PROBS_CUTOFF*d_CD_create_prefact * (QN.w - 1.0f));

		surf2Dwrite(wsh, s_probs, sizeof(int4)*i, j);
		//probability of Kuhn step shitt + probability of entanglement destruction by CD
		// + probability of entanglement creation by CD
	}

	//write updated chain conformation
	surf2Dwrite(QN, s_b_QN, 16 * i, j);
	surf2Dwrite(tcd, s_b_tCD, 4 * i, j);
	surf2Dwrite(t_cr, s_b_tcr, 4 * i, j);
}

__global__ void boundary1_kernel(scalar_chains* chain_heads, int *d_offset, float4 *d_new_strent){
	//calculate probabilities at the ends of arms
	int ii = blockIdx.x * blockDim.x + threadIdx.x;//arm index in ensemble
	int i = int(ii/d_narms);
	int arm = ii - i*d_narms;

	if (i >= dn_cha_per_call)
		return;
	float4 QNtail; //last strent
	float4 QNtailp;
	float4 new_strent = d_new_strent[i];

	int run_sum=0;

	for (int u=0; u<arm; u++){
		run_sum += d_z_max_arms[arm];
	}

	int tz = chain_heads[i].Z[arm];
	uint oft = d_offset[ii];

	int4 probs_z = make_int4(0,0,0,0);

	if (fetch_new_strent(tz - 1 + run_sum, oft))
		QNtail = new_strent;
	else
		QNtail = tex2D(t_a_QN, make_offset((tz - 1)+ run_sum, oft), i);

	if (tz == 1) {
		probs_z.y = (int)(PROBS_CUTOFF*__fdividef(1.0f, (dBe * dnk_arms[arm])));//Creation at the end by SD
	} else {
		if (QNtail.w == 1.0f) {//destruction by SD at the end
			if (fetch_new_strent(tz - 2 + run_sum, oft))
				QNtailp = new_strent;
			else
				QNtailp = tex2D(t_a_QN, make_offset(tz - 2 + run_sum, oft), i);
			float f1 = (tz == 2) ? QNtailp.w + 0.25f : 0.5f * QNtailp.w;
			probs_z.x = (int)(PROBS_CUTOFF*__fdividef(1.0f, f1 + 0.75f));
		} else {//creation by SD at the end
			probs_z.y = (int)(PROBS_CUTOFF*__fdividef(2.0f, dBe * (QNtail.w + 0.5f)));
		}
	}

	probs_z.w = d_CD_flag ? (int)(PROBS_CUTOFF*d_CD_create_prefact * (QNtail.w - 1.0f)) : 0;
	surf2Dwrite(probs_z, s_probs, sizeof(int4)*(tz - 1+run_sum), i);
}

template<int narms> __global__ void boundary2_kernel(scalar_chains* chain_heads, int *d_offset, float4 *d_new_strent, float *d_new_tau_CD){
	//calculate probabilities at the ends of chain
	int ii = blockIdx.x * blockDim.x + threadIdx.x;//arm index in ensemble
	int i = int(ii/narms);
	int arm = ii - i*narms;

	if (i >= dn_cha_per_call)
		return;
	float4 QNhead_arms[narms]; // first strent
	float4 new_strent = d_new_strent[i];
	float sumW = 0;
	int tz;
	int run_sum=0;
	for (int u=0; u<narms; u++){
		tz = chain_heads[i].Z[u];
		uint oft = d_offset[i*narms+u];

		float4 probs_z = make_float4(0.0f,0.0f,0.0f,0.0f);

		if (fetch_new_strent(0 + run_sum, oft))
			QNhead_arms[u] = new_strent;
		else
			QNhead_arms[u] = tex2D(t_a_QN, make_offset(0 + run_sum, oft), i);
		run_sum += d_z_max_arms[u];
	}
//	clock_t t2 = clock();
	//Shifts near branching point
	float4 QN2; //Q for next strent
	run_sum=0;
	float upsum1 = 0.0f;
	float upsum2 = 0.0f;
	float downsum1 = 0.0f;
	float downsum2 = 0.0f;
	float temp;

	run_sum=0;
	for (int u=0; u<arm; u++){
		run_sum += d_z_max_arms[u];
	}

	tz = chain_heads[i].Z[arm];
	if (tz>1){
		uint oft = d_offset[i*narms+arm];
		if (fetch_new_strent(1 + run_sum, oft))
			QN2 = new_strent;
		else
			QN2 = tex2D(t_a_QN, make_offset(1 + run_sum, oft), i);

		float Q2 = QN2.x * QN2.x + QN2.y * QN2.y + QN2.z * QN2.z;
		int4 probs_1 = make_int4(0,0,0,0);
		//backward shift
		if (QN2.w > 1.0f) { //N=1 mean that shift is not possible
			upsum1 = 0.0f;
			upsum2 = 0.0f;
			downsum1 = 0.0f;
			downsum2 = 0.0f;
			for (int q1=0; q1 < narms; q1++){
				if (chain_heads[i].Z[q1]>1){
					for (int q2=q1+1; q2 < narms; q2++){
						if (chain_heads[i].Z[q2]>1){
							temp = (QNhead_arms[q2].x - QNhead_arms[q1].x)*(QNhead_arms[q2].x - QNhead_arms[q1].x)+(QNhead_arms[q2].y - QNhead_arms[q1].y)*(QNhead_arms[q2].y - QNhead_arms[q1].y)+(QNhead_arms[q2].z - QNhead_arms[q1].z)*(QNhead_arms[q2].z - QNhead_arms[q1].z);
							upsum1 += __fdividef(0.75 * temp,QNhead_arms[q2].w * QNhead_arms[q1].w);
							upsum2 += __fdividef(0.75 * temp,(QNhead_arms[q2].w + (int)(q2==arm)) * (QNhead_arms[q1].w + (int)(q1==arm)));
						}
					}
					downsum1 += __fdividef(1.0f, QNhead_arms[q1].w);
					downsum2 += __fdividef(1.0f, QNhead_arms[q1].w + (int)(q1==arm));
				}
			}

			float sig2 = __fdividef(0.75f, QN2.w * (QN2.w - 1));
			float prefact1 = __fdividef(QNhead_arms[arm].w, (QNhead_arms[arm].w + 1));
			float prefact2 = (Q2 == 0.0f) ? 1.0f : __fdividef(QN2.w, (QN2.w - 1));
			float f1 = QNhead_arms[arm].w;
			float f2 = (tz == 2) ? 2.0f * QN2.w - 0.5f : QN2.w;
			float friction = __fdividef(2.0f, f1 + f2);
			probs_1.x = (int)(PROBS_CUTOFF*friction * __powf(__fdividef(prefact1 * prefact2 * downsum1,downsum2), 0.75f)* __expf(__fdividef(upsum1,downsum1) - __fdividef(upsum2,downsum2) - Q2 * sig2));
		}

		//forward shift
		if (QNhead_arms[arm].w > 1.0f) { //N=1 mean that shift is not possible
			upsum1 = 0.0f;
			upsum2 = 0.0f;
			downsum1 = 0.0f;
			downsum2 = 0.0f;
			for (int q1=0; q1 < narms; q1++){
				if (chain_heads[i].Z[q1]>1){
					for (int q2=q1+1; q2 < narms; q2++){
						if (chain_heads[i].Z[q2]>1){
							temp = (QNhead_arms[q2].x - QNhead_arms[q1].x)*(QNhead_arms[q2].x - QNhead_arms[q1].x)+(QNhead_arms[q2].y - QNhead_arms[q1].y)*(QNhead_arms[q2].y - QNhead_arms[q1].y)+(QNhead_arms[q2].z - QNhead_arms[q1].z)*(QNhead_arms[q2].z - QNhead_arms[q1].z);
							upsum1 += __fdividef(0.75 * temp,QNhead_arms[q2].w * QNhead_arms[q1].w);
							upsum2 += __fdividef(0.75 * temp,(QNhead_arms[q2].w - (int)(q2==arm)) * (QNhead_arms[q1].w - (int)(q1==arm)));
						}
					}
					downsum1 += __fdividef(1.0f, QNhead_arms[q1].w);
					downsum2 += __fdividef(1.0f, QNhead_arms[q1].w - (int)(q1==arm));
				}
			}

			float sig2 = __fdividef(0.75f, QN2.w * (QN2.w + 1));
			float prefact1 = __fdividef(QNhead_arms[arm].w, (QNhead_arms[arm].w - 1));
			float prefact2 = (Q2 == 0.0f) ? 1.0f : __fdividef(QN2.w, (QN2.w + 1));
			float f1 = QNhead_arms[arm].w;
			float f2 = (tz == 2) ? 2.0f * QN2.w - 0.5f : QN2.w;
			float friction = __fdividef(2.0f, f1 + f2);
			probs_1.y = (int)(PROBS_CUTOFF*friction * __powf(__fdividef(prefact1 * prefact2 * downsum1,downsum2), 0.75f)* __expf(__fdividef(upsum1,downsum1) - __fdividef(upsum2,downsum2) + Q2 * sig2));
		}
		
		float tcd = 0.0f;

		if (d_CD_flag) { //If constraint dynamics is enabled
			tcd = fetch_new_strent(run_sum, oft) ? d_new_tau_CD[i] : tex2D(t_a_tCD, make_offset(run_sum, oft), i);
		}

		probs_1.z = d_CD_flag ? (int)(PROBS_CUTOFF*tcd) : 0;
		probs_1.w = d_CD_flag ? (int)(PROBS_CUTOFF*d_CD_create_prefact * (QNhead_arms[arm].w - 1.0f)) : 0;
		surf2Dwrite(probs_1, s_probs, sizeof(int4)*run_sum, i);
	}
}

__global__ void scan_kernel(scalar_chains* chain_heads, int *rand_used, int* found_index,  int* found_shift, float* add_rand) {
	extern __shared__ int s[];
	//Calculate kernel index
	int i = blockIdx.x * blockDim.x + threadIdx.x;//strent index
	int j = blockIdx.y * blockDim.y + threadIdx.y;//chain index
	//Check if kernel index is outside boundaries

	int4 temp = make_int4(0, 0, 0, 0);

	int read_flag;
	surf2Dread(&read_flag, s_arm_index, sizeof(int)*i, j);

	if (read_flag){
		surf2Dread(&temp, s_probs, sizeof(int4)*i, j);
	}

	//parallel scan in s
	int var = d_CD_flag ? temp.x + temp.y + temp.z + temp.w : temp.x + temp.y;
	
	//warp scan
	for (int d = 1; d<32; d <<= 1) {
		int var2 = __shfl_up(var, d);
		if (i % 32 >= d)
			var += var2;
	}

	if (i % 32 == 31) s[i / 32] = var;
	__syncthreads();

	//scan of warp sums
	if (i < 32) {
		int var2 = 0.0f;
		if (i < blockDim.x / 32)
			var2 = s[i];
		for (int d = 1; d<32; d <<= 1) {
			float var3 = __shfl_up(var2, d);
			if (i % 32 >= d) var2 += var3;
		}
		if (i < blockDim.x / 32) s[i] = var2;
	}
	__syncthreads();

	if (i >= 32) var += s[i / 32 - 1];
	
	__syncthreads();
	
	s[i] = var;

	__syncthreads();
	surf2Dwrite((float)(s[i])/PROBS_CUTOFF, s_sum_W_sorted, sizeof(float)*i, j);

	//search
	float ran = tex2D(t_uniformrand, rand_used[j], j);
	int x = ceil((float)s[d_z_max-1]*ran);
	int left = (i==0)? 0 : s[i - 1];
	int right = s[i];

	bool xFound = (left < x) && (x <= left + temp.x);
	bool yFound = (left + temp.x < x) && (x <= left + temp.x + temp.y);
	bool zFound = (left + temp.x + temp.y < x) && (x <= left + temp.x + temp.y + temp.z);
	bool wFound = (left + temp.x + temp.y + temp.z < x) && (x <= left + temp.x + temp.y + temp.z + temp.w);

	if (xFound || yFound || zFound || wFound) {
		found_index[j] = i;
		if (xFound)			found_shift[j] = 0;
		else if (yFound)	found_shift[j] = 1;
		else if (zFound)	found_shift[j] = 2; //destruction by CD
		else if (wFound) {
			found_shift[j] = 4; //creation by CD
			add_rand[j] = (float)(x - left - temp.x - temp.y - temp.z) / (float)temp.w;
		}
	}
}

//Add new value w to k-th level of correlator corr for chain i
__device__ void corr_add(corr_device gpu_corr, float4 w, int k, int i, int type) {
	int s = *(gpu_corr.d_numcorrelators);
	//s is the last correlator level
	if (k == s)
		return;

	int dm = *(gpu_corr.d_dmin);
	int p = *(gpu_corr.d_correlator_size);
	int corr_aver_s = *(gpu_corr.d_correlator_aver_size);

	//extract 3D array pointers and pitches
	char* shift_ptr = (char *) (gpu_corr.d_shift.ptr);
	size_t shift_pitch = gpu_corr.d_shift.pitch;
	size_t shift_slicePitch = shift_pitch * s;
	char* shift_slice = shift_ptr + i * shift_slicePitch;
	float4* shift = (float4*) (shift_slice + k * shift_pitch);

	char* correlation_ptr = (char *) (gpu_corr.d_correlation.ptr);
	size_t correlation_pitch = gpu_corr.d_correlation.pitch;
	size_t correlation_slicePitch = correlation_pitch * s;
	char* correlation_slice = correlation_ptr + i * correlation_slicePitch;
	float4* correlation = (float4*) (correlation_slice + k * correlation_pitch);

	char* ncorrelation_ptr = (char *) (gpu_corr.d_ncorrelation.ptr);
	size_t ncorrelation_pitch = gpu_corr.d_ncorrelation.pitch;
	size_t ncorrelation_slicePitch = ncorrelation_pitch * s;
	char* ncorrelation_slice = ncorrelation_ptr + i * ncorrelation_slicePitch;
	float* ncorrelation = (float*) (ncorrelation_slice + k * ncorrelation_pitch);

	//Extract 2D array pointers and pitches
	float4* accumulator = (float4*) ((char*) gpu_corr.d_accumulator + i * gpu_corr.d_accumulator_pitch);
	int* naccumulator = (int*) ((char*) gpu_corr.d_naccumulator + i * gpu_corr.d_naccumulator_pitch);
	int* insertindex = (int*) ((char*) gpu_corr.d_insertindex + i * gpu_corr.d_insertindex_pitch);

	//update maximum attained correlator level
	if (k > gpu_corr.d_kmax[i])
		gpu_corr.d_kmax[i] = k;

	//Write new value to the shift array
	shift[insertindex[k]] = w;

	//Update average value
	if (k == 0){
		gpu_corr.d_accval[i] += w;
	}

	//Add to accumulator and send to the next level, if needed
	if (type==0 || (type==1 && naccumulator[k]==0)){
		accumulator[k] += w;
	}
	naccumulator[k]++;
	if (naccumulator[k] == corr_aver_s) {
		//Calling next correlator
		if (type==0)	corr_add(gpu_corr, (accumulator[k]) / float(corr_aver_s), k + 1, i, type);
		if (type==1)	corr_add(gpu_corr, (accumulator[k]), k + 1, i, type);
		accumulator[k] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
		naccumulator[k] = 0;
	}

	//Update correlation results
	int ind1 = insertindex[k];
	float4 temp_shift_1 = shift[ind1];//cache frequently used value in register
	float4 temp_shift_2;
	if (k == 0) {
		int ind2 = ind1;
		for (int j = 0; j < p; ++j) {
			temp_shift_2 = shift[ind2];
			if (temp_shift_2.x != 0.0f || temp_shift_2.y != 0.0f || temp_shift_2.z != 0.0f) {
				if (type==0)	correlation[j] += temp_shift_1 * temp_shift_2;
				if (type==1)	correlation[j] += (temp_shift_1 - temp_shift_2)*(temp_shift_1 - temp_shift_2);
				ncorrelation[j] += 1.0f;
			}
			--ind2;
			if (ind2 < 0)
				ind2 += p;
		}
	} else {
		int ind2 = ind1 - dm;
		for (int j = dm; j < p; ++j) {
			if (ind2 < 0)
				ind2 += p;
			temp_shift_2 = shift[ind2];
			if (temp_shift_2.x != 0.0f || temp_shift_2.y != 0.0f || temp_shift_2.z != 0.0f) {
				if (type==0)	correlation[j] += temp_shift_1 * temp_shift_2;
				if (type==1)	correlation[j] += (temp_shift_1 - temp_shift_2)*(temp_shift_1 - temp_shift_2);
				ncorrelation[j] += 1.0f;
			}
			--ind2;
		}
	}
	insertindex[k]++;
	if (insertindex[k] == p)
		insertindex[k] = 0;
}

__global__ void update_correlator(corr_device gpu_corr, int n, int type){
	int i = blockIdx.x * blockDim.x + threadIdx.x; //Chain index
	if (i >= dn_cha_per_call)
		return;
	float4 stress;
	for (int j=0; j<n; j++){
		stress = tex2D(t_corr, i, j);
		if (stress.w != -1.0f){
			corr_add(gpu_corr, stress, 0, i, type); //add new value to the correlator
		}
	}
}

__global__ __launch_bounds__(tpb_chain_kernel) void flow_stress(corr_device gpu_corr, int n, float4* stress_average, int nc){
	int i = blockIdx.x * blockDim.x + threadIdx.x; //Chain index
	if (i >= dn_cha_per_call)
		return;
	float4 stress;
	for (int j=0; j<n; j++){
		stress = tex2D(t_corr, i, j);
		if (stress.w != -1.0f){
			stress_average[(int)stress.w * nc + i].x = stress.x;
			stress_average[(int)stress.w * nc + i].y = stress.y;
			stress_average[(int)stress.w * nc + i].z = stress.z;
			//atomicAdd(&(stress_average[(int)stress.w].x),stress.x);
			//atomicAdd(&(stress_average[(int)stress.w].y),stress.y);
			//atomicAdd(&(stress_average[(int)stress.w].z),stress.z);
		}
	}
}

template<int type> __global__ __launch_bounds__(tpb_chain_kernel) void chain_control_kernel(
	scalar_chains* chain_heads, float *tdt, float *reach_flag,
	float next_sync_time, int *d_offset, float4 *d_new_strent,
	int *d_write_time, int correlator_type, int stress_index)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;//chain index
	if (i >= dn_cha_per_call)
		return;

	float4 sum_stress = make_float4(0.0f, 0.0f, 0.0f, -1.0f);
	surf2Dwrite(sum_stress, s_corr, sizeof(float4) * i, stress_index); //Write stress value to the stack

	if (reach_flag[i] != 0) {
		return;
	}

	if (((chain_heads[i].time >= next_sync_time) && (d_universal_time + next_sync_time <= d_write_time[i] * d_correlator_res)) || (chain_heads[i].stall_flag != 0)) {
		reach_flag[i] = 1;
		//printf("\nreach_flag[i]=%f\ttime=%f", reach_flag[i], d_universal_time + chain_heads[i].time);
		chain_heads[i].time -= next_sync_time;
		tdt[i] = 0.0f;
		for (int u = 0; u<d_narms; u++) {
			d_offset[i*d_narms + u] = offset_code(0xffff, +1);
		}
		return;
	}

	float4 new_strent = d_new_strent[i];

	//check for correlator
	if (d_universal_time + chain_heads[i].time > d_write_time[i] * d_correlator_res) { //TODO add d_correlator_time to gpu_chain_heads
		if (correlator_type == 0) {//stress calc
			int run_sum_ = 0;
			for (int arm_ = 0; arm_ < d_narms; arm_++) {
				int tz_ = chain_heads[i].Z[arm_];
				for (int j = 0; j < tz_; j++) {
					float4 QN1;
					if (fetch_new_strent(j + run_sum_, d_offset[i*d_narms + arm_]))
						QN1 = new_strent;
					else
						QN1 = tex2D(t_a_QN, make_offset(j + run_sum_, d_offset[i*d_narms + arm_]), i);

					sum_stress.x -= __fdividef(3.0f * QN1.x * QN1.y, QN1.w);
					sum_stress.y -= __fdividef(3.0f * QN1.y * QN1.z, QN1.w);
					sum_stress.z -= __fdividef(3.0f * QN1.x * QN1.z, QN1.w);
				}
				run_sum_ += d_z_max_arms[arm_];
			}
			sum_stress.w = 1.0f;
			surf2Dwrite(sum_stress, s_corr, sizeof(float4) * i, stress_index); //Write stress value to the stack
		}

		//Update counter
		d_write_time[i]++;
	}
}

template<int type> __global__ __launch_bounds__(tpb_chain_kernel) void chain_kernel(
		scalar_chains* chain_heads, float *tdt, float *reach_flag, 
		float next_sync_time, int *d_offset, float4 *d_new_strent, 
		float *d_new_tau_CD, float* d_new_cr_time, int *d_write_time, int correlator_type,
		int *rand_used, int *tau_CD_used_CD, int *tau_CD_used_SD,
		int* found_index, int* found_shift, float* add_rand)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;//chain index

	if (i >= dn_cha_per_call)
		return;

	surf1Dwrite(0.0f,s_ft,i*sizeof(float));
	if (reach_flag[i]!=0) {
		return;
	}

	//check again to stop if necessary
	if (d_universal_time + chain_heads[i].time > d_write_time[i] * d_correlator_res) {
		for (int u = 0; u<d_narms; u++) {
			d_offset[i*d_narms + u] = offset_code(0xffff, +1);
		}
		return;
	}

	float olddt;
	if (type == 1) olddt = tdt[i];
	float4 new_strent = d_new_strent[i];

	float sumW; // sum of probabilities
	surf2Dread(&sumW, s_sum_W_sorted, sizeof(float)*(d_z_max-1), i);

	//decide the timestep
	tdt[i] = __fdividef(1.0f, sumW);

	// error handling
	if (tdt[i] == 0.0f)	chain_heads[i].stall_flag = 1;
	if (isnan(tdt[i]))	chain_heads[i].stall_flag = 2;
	if (isinf(tdt[i]))	chain_heads[i].stall_flag = 3;
	
	chain_heads[i].time += tdt[i];//update time
	rand_used[i]++;

	int j = found_index[i];
	int k = found_shift[i];
	
	int arm=0;
	int run_sum=0;
	for (arm=0; j>=run_sum+d_z_max_arms[arm]; arm++){
		run_sum+=d_z_max_arms[arm];
	}
	int jj = j-run_sum;
	int ii = i*d_narms+arm;

	//setup local variables
	int tz = chain_heads[i].Z[arm];
	uint oft = d_offset[ii];

	if (jj == tz - 1) {//boundary cases
		if (k == 0) {//destruction by SD at the end
			k = 5;
			jj--;
			j--;
		}
		else if (k == 1) {//creation by SD at the end
			k = 3;
		}
	}

	float4 QN1 = fetch_new_strent(j, oft) ? new_strent : tex2D(t_a_QN, make_offset(j, oft), i);
	float4 QN2 = fetch_new_strent(j+1, oft) ? new_strent : tex2D(t_a_QN, make_offset(j + 1, oft), i);

	if (type == 1) {
		QN1 = kappa(QN1, olddt);
		QN2 = kappa(QN2, olddt);
	}

	//Apply chosen process (shuffling of Kuhn step, destruction or creation of entanglement)
	if (k==0 || k==1) {//Shuffling
		if (k==0) { //shuffling left
			QN1.w = QN1.w + 1;
			QN2.w = QN2.w - 1;
			//printf("\nShift left: arm %i\tstrent%i", arm, jj);
		} else { //shuffling right
			QN1.w = QN1.w - 1;
			QN2.w = QN2.w + 1;
			//printf("\nShift right: arm %i\tstrent%i", arm, jj);
		}

		if (jj == 0) {//shuffling invoving branch-point
			float sumNinv = 0.0f;
			float4 temp_;
			int run_sum_ = 0;
			for (int arm_ = 0; arm_<d_narms; arm_++) {
				int tz_ = chain_heads[i].Z[arm_];
				if ((tz_>1) && (arm_ != arm)) {//entangled arms
					if (fetch_new_strent(0 + run_sum_, d_offset[i*d_narms + arm_]))
						temp_ = new_strent;
					else
						temp_ = tex2D(t_a_QN, make_offset(0 + run_sum_, d_offset[i*d_narms + arm_]), i);
					sumNinv += 1 / temp_.w;
				}
				run_sum_ += d_z_max_arms[arm_];
			}
			sumNinv += 1 / QN1.w;
			float4 deltaQ = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
			if (k == 0) {//shift left
				deltaQ = QN1 / (-QN1.w*(QN1.w - 1)*sumNinv);
			}
			else {
				deltaQ = QN1 / (QN1.w*(QN1.w + 1)*sumNinv);
			}
			deltaQ.w = 0.0f;

			run_sum_ = 0;
			for (int arm_ = 0; arm_ < d_narms; arm_++) {
				int tz_ = chain_heads[i].Z[arm_];
				if ((tz_ > 1) && (arm_ != arm)) {//entangled arms
					if (fetch_new_strent(0 + run_sum_, d_offset[i*d_narms + arm_]))
						temp_ = new_strent;
					else
						temp_ = tex2D(t_a_QN, make_offset(0 + run_sum_, d_offset[i*d_narms + arm_]), i);
					
					temp_ = temp_ - deltaQ;
					
					surf2Dwrite(temp_, s_b_QN, 16 * run_sum_, i);
				}
				run_sum_ += d_z_max_arms[arm_];
			}
			QN1 = QN1 - deltaQ;
		}
		surf2Dwrite(QN1, s_b_QN, 16 * j, i);
		surf2Dwrite(QN2, s_b_QN, 16 * (j + 1), i);
		d_offset[ii] = offset_code(0xffff, +1);
	}
	else if (k == 2 || k == 5) {
		//printf("\nDestruction of entanglement: arm %i\tstrent%i", arm, jj);
		// Destruction by sliding dynamics
		chain_heads[i].Z[arm]--;  //decrease number of strands as entanglement is destroyed

		float4 temp = make_float4(QN1.x + QN2.x, QN1.y + QN2.y, QN1.z + QN2.z, QN1.w + QN2.w); //temporary variable for new strand
		if (jj == tz - 2){
			temp = make_float4(0.0f, 0.0f, 0.0f, QN1.w + QN2.w);
		}

		float4 deltaQ;
		if (chain_heads[i].Z[arm]==1) { //arm go unentangled as a result of entanglement destruction
			float4 temp_;
			int run_sum_ = 0;
			bool unent = true; //if all arms are unentangled -> set temp as (0,0,0)
			float sumNinv = 0.0f;
			for (int arm_ = 0; arm_<d_narms; arm_++) {
				int tz_ = chain_heads[i].Z[arm_];
				if ((tz_>1) && (arm_ != arm)) {//entangled arms
					unent = false;

					if (fetch_new_strent(0 + run_sum_, d_offset[i*d_narms + arm_]))
						temp_ = new_strent;
					else
						temp_ = tex2D(t_a_QN, make_offset(0 + run_sum_, d_offset[i*d_narms + arm_]), i);

					sumNinv += 1 / temp_.w;
				}
				run_sum_ += d_z_max_arms[arm_];
			}

			deltaQ = QN1 / (QN1.w*sumNinv);
			deltaQ.w = 0.0f;

			run_sum_ = 0;
			for (int arm_ = 0; arm_ < d_narms; arm_++) {
				int tz_ = chain_heads[i].Z[arm_];
				if ((tz_ > 1) && (arm_ != arm)) {//entangled arms
					if (fetch_new_strent(0 + run_sum_, d_offset[i*d_narms + arm_]))
						temp_ = new_strent;
					else
						temp_ = tex2D(t_a_QN, make_offset(0 + run_sum_, d_offset[i*d_narms + arm_]), i);

					temp_ = temp_ + deltaQ;

					surf2Dwrite(temp_, s_b_QN, 16 * run_sum_, i);
				}
				run_sum_ += d_z_max_arms[arm_];
			}

		}
		else if (jj == 0) {
			float4 temp_;
			int run_sum_ = 0;
			bool unent = true; //if all arms are unentangled -> set temp as (0,0,0)
			float sumNinv = 0.0f;
			float sumNinvnew = 0.0f;
			for (int arm_ = 0; arm_<d_narms; arm_++) {
				int tz_ = chain_heads[i].Z[arm_];
				if ((tz_>1) && (arm_ != arm)) {//entangled arms
					unent = false;

					if (fetch_new_strent(0 + run_sum_, d_offset[i*d_narms + arm_]))
						temp_ = new_strent;
					else
						temp_ = tex2D(t_a_QN, make_offset(0 + run_sum_, d_offset[i*d_narms + arm_]), i);

					sumNinv += 1 / temp_.w;
					sumNinvnew += 1 / temp_.w;
				}
				run_sum_ += d_z_max_arms[arm_];
			}
			sumNinvnew += __fdividef(1.0f, QN1.w + QN2.w);

			deltaQ.x = __fdividef(__fdividef(QN2.x, QN1.w + QN2.w) - __fdividef(QN1.x * QN2.w, QN1.w * (QN1.w + QN2.w)), -sumNinvnew);
			deltaQ.y = __fdividef(__fdividef(QN2.y, QN1.w + QN2.w) - __fdividef(QN1.y * QN2.w, QN1.w * (QN1.w + QN2.w)), -sumNinvnew);
			deltaQ.z = __fdividef(__fdividef(QN2.z, QN1.w + QN2.w) - __fdividef(QN1.z * QN2.w, QN1.w * (QN1.w + QN2.w)), -sumNinvnew);
			deltaQ.w = 0.0f;

			run_sum_ = 0;
			for (int arm_ = 0; arm_ < d_narms; arm_++) {
				int tz_ = chain_heads[i].Z[arm_];
				if ((tz_ > 1) && (arm_ != arm)) {//entangled arms
					if (fetch_new_strent(0 + run_sum_, d_offset[i*d_narms + arm_]))
						temp_ = new_strent;
					else
						temp_ = tex2D(t_a_QN, make_offset(0 + run_sum_, d_offset[i*d_narms + arm_]), i);

					temp_ = temp_ + deltaQ;
					surf2Dwrite(temp_, s_b_QN, 16 * run_sum_, i);
				}
				run_sum_ += d_z_max_arms[arm_];
			}

			temp.x += deltaQ.x;
			temp.y += deltaQ.y;
			temp.z += deltaQ.z;
		}

		surf2Dwrite(temp, s_b_QN, 16 * (jj + 1 + run_sum), i);
		d_offset[ii] = offset_code(jj + run_sum, +1);

		if (k==5){
			float cr_time;
			if (fetch_new_strent(jj + run_sum, oft)) {
				cr_time = d_new_cr_time[i];
			}
			else {
				cr_time = tex2D(t_a_tcr, make_offset(jj + run_sum, oft), i);
			}
			if (cr_time != 0) {
				surf1Dwrite(log10f(d_universal_time + chain_heads[i].time - cr_time) + 10, s_ft, i * sizeof(float));
			}
		}
	} 
	else if (k==3) {//  Creation by SD
		//printf("\nCreation by SD: arm %i\tstrent%i", arm, jj);
		float4 temp = tex2D(t_taucd_gauss_rand_SD, tau_CD_used_SD[i], i);
		tau_CD_used_SD[i]++;
		chain_heads[i].Z[arm]++;
		d_new_tau_CD[i] = temp.w;
		d_new_cr_time[i] = d_universal_time + chain_heads[i].time;

		temp.w = QN1.w - 1.0f;

		float sigma = __fsqrt_rn(__fdividef(temp.w, 3.0f));
		temp.x *= sigma;
		temp.y *= sigma;
		temp.z *= sigma;
		if (tz==1){
			float4 temp_;
			int run_sum_ = 0;
			bool unent=true; //if all arms are unentangled -> set temp as (0,0,0)
			float sumNinv = 0.0f;
			for (int arm_=0; arm_<d_narms; arm_++){
				int tz_ = chain_heads[i].Z[arm_];
				if ((tz_>1) && (arm_!=arm)){//entangled arms
					unent=false;

					if (fetch_new_strent(0 + run_sum_, d_offset[i*d_narms + arm_]))
						temp_ = new_strent;
					else
						temp_ = tex2D(t_a_QN, make_offset(0 + run_sum_, d_offset[i*d_narms + arm_]), i);

					sumNinv += 1/temp_.w;
				}
				run_sum_ += d_z_max_arms[arm_];
			}

			float4 deltaQ = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
			deltaQ = temp / (-temp.w*sumNinv);
			deltaQ.w = 0.0f;


			run_sum_ = 0;
			for (int arm_ = 0; arm_ < d_narms; arm_++) {
				int tz_ = chain_heads[i].Z[arm_];
				if ((tz_ > 1) && (arm_ != arm)) {//entangled arms
					if (fetch_new_strent(0 + run_sum_, d_offset[i*d_narms + arm_]))
						temp_ = new_strent;
					else
						temp_ = tex2D(t_a_QN, make_offset(0 + run_sum_, d_offset[i*d_narms + arm_]), i);

					temp_ = temp_ + deltaQ;
					surf2Dwrite(temp_, s_b_QN, 16 * run_sum_, i);
				}
				run_sum_ += d_z_max_arms[arm_];
			}

			if(unent){
				temp.x = 0.0f;
				temp.y = 0.0f;
				temp.z = 0.0f;
			}
		}
		surf2Dwrite(make_float4(0.0f, 0.0f, 0.0f, 1.0f), s_b_QN, 16 * j, i);
		d_offset[ii] = offset_code(j, -1);
		d_new_strent[i] = temp;
	}
	else if (k == 4) {//Creation by CD
		//printf("\nCreation by CD: arm %i\tstrent%i",arm,jj);
		float4 temp = tex2D(t_taucd_gauss_rand_CD, tau_CD_used_CD[i], i);
		tau_CD_used_CD[i]++;
		chain_heads[i].Z[arm]++;
		d_new_tau_CD[i] = temp.w;
		d_new_cr_time[i] = d_universal_time + chain_heads[i].time;
		//d_new_cr_time[i] = 0.0f;
		float newn = floorf(0.5f + add_rand[i] * (QN1.w - 2.0f)) + 1.0f;
		temp.w = newn;
		float sigma = __fsqrt_rn(__fdividef(newn * (QN1.w - newn), 3.0f * QN1.w));
		if (jj == tz - 1) {
			sigma = __fsqrt_rn(__fdividef(newn, 3.0f));
		}
		float ration = __fdividef(newn, QN1.w);

		float4 deltaQ = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
		if (jj == 0) {//new strand near branch point
			float4 temp_;
			int run_sum_ = 0;
			bool unent = true; //if all arms are unentangled -> set temp as (0,0,0)
			float sumNinv = 0.0f;
			float sumNinvnew = 0.0f;
			for (int arm_ = 0; arm_<d_narms; arm_++) {
				int tz_ = chain_heads[i].Z[arm_];
				if ((tz_>1) && (arm_ != arm)) {//entangled arms
					unent = false;

					if (fetch_new_strent(0 + run_sum_, d_offset[i*d_narms + arm_]))
						temp_ = new_strent;
					else
						temp_ = tex2D(t_a_QN, make_offset(0 + run_sum_, d_offset[i*d_narms + arm_]), i);

					sumNinv += 1 / temp_.w;
					sumNinvnew += 1 / temp_.w;
				}
				run_sum_ += d_z_max_arms[arm_];
			}
			sumNinvnew += 1 / newn;
			sumNinv += 1 / QN1.w;
			sigma = __fsqrt_rn(__fdividef(newn * (QN1.w - newn) * sumNinvnew, 3.0f * QN1.w * sumNinv));
			float factor = (1.0f - __fdividef(1.0f, newn * sumNinvnew));

			deltaQ.x = __fdividef(-temp.x * sigma, newn * sumNinvnew);
			deltaQ.y = __fdividef(-temp.y * sigma, newn * sumNinvnew);
			deltaQ.z = __fdividef(-temp.z * sigma, newn * sumNinvnew);
			deltaQ.w = 0.0f;

			sigma *= factor;
			run_sum_ = 0;
			for (int arm_ = 0; arm_ < d_narms; arm_++) {
				int tz_ = chain_heads[i].Z[arm_];
				if ((tz_ > 1) && (arm_ != arm)) {//entangled arms
					if (fetch_new_strent(0 + run_sum_, d_offset[i*d_narms + arm_]))
						temp_ = new_strent;
					else
						temp_ = tex2D(t_a_QN, make_offset(0 + run_sum_, d_offset[i*d_narms + arm_]), i);

					temp_ = temp_ + deltaQ;
					surf2Dwrite(temp_, s_b_QN, 16 * run_sum_, i);
				}
				run_sum_ += d_z_max_arms[arm_];
			}
		}

		temp.x *= sigma;
		temp.y *= sigma;
		temp.z *= sigma;
		temp.x += QN1.x * ration;
		temp.y += QN1.y * ration;
		temp.z += QN1.z * ration;

		// if (unent_cr)
		// 	printf("\nQ=%f\t%f\t%f\tN=%f", temp.x, temp.y, temp.z, temp.w);

		surf2Dwrite(make_float4(QN1.x - temp.x, QN1.y - temp.y, QN1.z - temp.z, QN1.w - newn), s_b_QN, 16 * j, i);
		if (jj == 0) {
			surf2Dwrite(make_float4(QN1.x - temp.x + deltaQ.x, QN1.y - temp.y + deltaQ.y, QN1.z - temp.z + deltaQ.z, QN1.w - newn), s_b_QN, 16 * j, i);
		}
		if (jj == tz - 1) {
			surf2Dwrite(make_float4(0.0f, 0.0f, 0.0f, QN1.w - newn), s_b_QN, 16 * j, i);
		}

		d_offset[ii] = offset_code(j, -1);

		d_new_strent[i] = temp;
	}

	for (int u = 0; u<d_narms; u++) {
		if (u != arm)
			d_offset[i*d_narms + u] = offset_code(0xffff, +1);
	}
	return;
}

__global__ __launch_bounds__(tpb_chain_kernel) void stress_calc(scalar_chains* chain_heads, float *tdt, int *d_offset, float4 *d_new_strent, float4* QN, int size) {//stress calculation
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= dn_cha_per_call)
		return;
	int tz = chain_heads[i].Z[1];
	uint oft = d_offset[i];
	float olddt = tdt[i];
	float4 new_strent = d_new_strent[i];

	float4 sum_stress = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	float4 sum_stress2 = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	for (int j = 0; j < tz; j++) {
		float4 QN1 = tex2D(t_a_QN, make_offset(j, oft), i);
		if (fetch_new_strent(j, oft))
			QN1 = new_strent;
		QN1 = kappa(QN1, olddt);
		sum_stress.x -= __fdividef(3.0f * QN1.x * QN1.x, QN1.w);
		sum_stress.y -= __fdividef(3.0f * QN1.y * QN1.y, QN1.w);
		sum_stress.z -= __fdividef(3.0f * QN1.z * QN1.z, QN1.w);
		sum_stress.w -= __fdividef(3.0f * QN1.x * QN1.y, QN1.w);
		sum_stress2.x -= __fdividef(3.0f * QN1.y * QN1.z, QN1.w);
		sum_stress2.y -= __fdividef(3.0f * QN1.x * QN1.z, QN1.w);
		sum_stress2.z += __fsqrt_rn(QN1.x * QN1.x + QN1.y * QN1.y + QN1.z * QN1.z);
	}
	sum_stress2.w = float(tz);
	surf1Dwrite(sum_stress, s_stress, 32 * i);
	surf1Dwrite(sum_stress2, s_stress, 32 * i + 16);

}

#endif
