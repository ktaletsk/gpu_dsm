#include "hip/hip_runtime.h"
// Copyright 2014 Marat Andreev
// 
// This file is part of gpu_dsm.
// 
// gpu_dsm is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// at your option) any later version.
// 
// gpu_dsm is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with gpu_dsm.  If not, see <http://www.gnu.org/licenses/>.

//  #include <hip/hip_runtime.h>
#include "gpu_random.h"
#include "random.h"
#include "pcd_tau.h"
#include "ensemble.h"
#include "gamma.cu"
#include <fstream>

extern p_cd *pcd;
extern bool PD_flag;
// #include <iostream>


#if defined(_MSC_VER)
#define uint unsigned int
#endif

#include "cudautil.h"
#include "cuda_call.h"      
#include "textures_surfaces.h"

//CD constants
__constant__ float d_At, d_Ct, d_Dt, d_Adt, d_Bdt, d_Cdt, d_Ddt;
__constant__ float d_g, d_alpha, d_tau_0, d_tau_max, d_tau_d, d_tau_d_inv;
__constant__ bool d_PD_flag;

void gpu_ran_init (){
	cout << "preparing GPU random number generator parameters..\n";

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_g), &(pcd->g), sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_alpha), &(pcd->alpha), sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tau_0), &(pcd->tau_0), sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tau_max), &(pcd->tau_max), sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tau_d), &(pcd->tau_d), sizeof(float)));
	float cdtemp = 1.0f / pcd->tau_d;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tau_d_inv), &(cdtemp), sizeof(float)));

	cdtemp = 1.0f / pcd->At;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_At), &cdtemp, sizeof(float)));
	cdtemp = powf(pcd->tau_0, pcd->alpha);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_Dt), &cdtemp, sizeof(float)));
	cdtemp = -1.0f / pcd->alpha;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_Ct), &cdtemp, sizeof(float)));
	cdtemp = pcd->normdt / pcd->Adt;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_Adt), &cdtemp, sizeof(float)));
	cdtemp = pcd->Bdt / pcd->normdt;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_Bdt), &cdtemp, sizeof(float)));
	cdtemp = -1.0f / (pcd->alpha - 1.0f);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_Cdt), &cdtemp, sizeof(float)));
	cdtemp = powf(pcd->tau_0, pcd->alpha - 1.0f);
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_Ddt), &(cdtemp), sizeof(float)));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_PD_flag), &PD_flag, sizeof(int)));

	if(PD_flag){
		double a, b, mp, Mk;
		ifstream in2;
		in2.open("polydisp.dat");
		in2 >> a;
		in2 >> b;
		in2 >> mp;
		in2 >> Mk;
		gamma(a,b);
		hipArray* d_gamma_table;
		hipChannelFormatDesc channelDesc1 = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
		hipMallocArray(&(d_gamma_table), &channelDesc1, uniformrandom_count, table_size, hipArraySurfaceLoadStore);
		//TODO hipMallocArray(&d_sum_W, &channelDesc1, z_max, rsz, hipArraySurfaceLoadStore);
		//
	}
	cout << "device random number generator parameters done\n";
	}

//
__global__ __launch_bounds__(ran_tpd) void fill_surface_rand (gpu_Ran *state,int n,int count ){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	float tmp;
	if (i<n){
		hiprandState localState = state[i];
		for (int j=0; j<count;j++){
			tmp=hiprand_uniform (&localState);
			surf2Dwrite(tmp,rand_buffer,4*j,i);
		}
		state[i] = localState;
	}
}

//
__global__ __launch_bounds__(ran_tpd) void array_seed (gpu_Ran *gr,int sz,int seed_offset){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	if (i<sz) hiprand_init(seed_offset, i, 0, &gr[i]);
}

//
void gr_array_seed (gpu_Ran *gr,int sz, int seed_offset){      
	array_seed<<<(sz+ ran_tpd-1)/ ran_tpd, ran_tpd>>>(gr,sz, seed_offset);
	CUT_CHECK_ERROR("kernel execution failed");
 	hipDeviceSynchronize();
}

//
void gr_fill_surface_uniformrand(gpu_Ran *gr,int sz,int count , hipArray*  d_uniformrand){
	cudaBindSurfaceToArray(rand_buffer, d_uniformrand);
	CUT_CHECK_ERROR("kernel execution failed");
	fill_surface_rand<<<(sz+ ran_tpd-1)/ ran_tpd, ran_tpd>>>(gr,sz,count);
	CUT_CHECK_ERROR("kernel execution failed");
	hipDeviceSynchronize();
}

//lifetime generation from uniform random number p
__device__ __forceinline__ float d_tau_CD_f_d_t(float p, float d_Adt, float d_Bdt, float d_Cdt, float d_Ddt, float d_tau_d_inv) {
	return p < d_Bdt ? __powf(p * d_Adt + d_Ddt, d_Cdt) : d_tau_d_inv;
}
__device__ __forceinline__ float d_tau_CD_f_t(float p, float d_At, float d_Ct, float d_Dt, float d_tau_d_inv, float d_g) {
	return p < 1.0f - d_g ? __powf(p * d_At + d_Dt, d_Ct) : d_tau_d_inv;
}

//
__global__ __launch_bounds__(ran_tpd) void fill_surface_taucd_gauss_rand (gpu_Ran *state, int n, int count, bool SDCD_toggle){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	float4 tmp;
	float g=0.0f;
	float2 g2;
	if (i<n){
		hiprandState localState = state[i];
		for (int j=0; j<count;j++){
			//Pcd generation for new entanglements

			//Pick a uniform distributed random number
			tmp.x=hiprand_uniform (&localState);

			if (d_PD_flag){
				tmp.y=hiprand_uniform (&localState); //For picking random molecular weight
				//TODO tmp.y=READ_TEXTURE[t_gamma_table[hiprand_uniform (&localState)/step]
				//TODO Calculate Pcd parameters (How?)
				if (SDCD_toggle == true)
					tmp.w = d_tau_CD_f_t(tmp.x, d_At, d_Ct, d_Dt, d_tau_d_inv, d_g);
				else
					tmp.w = d_tau_CD_f_d_t(tmp.x, d_Adt, d_Bdt, d_Cdt, d_Ddt, d_tau_d_inv);
				//TODO Substitute with calculated Pcd values
			}
			else{
				if (SDCD_toggle == true)
					tmp.w = d_tau_CD_f_t(tmp.x, d_At, d_Ct, d_Dt, d_tau_d_inv, d_g);
				else
					tmp.w = d_tau_CD_f_d_t(tmp.x, d_Adt, d_Bdt, d_Cdt, d_Ddt, d_tau_d_inv);
			}

			//Q vector generation for new entanglements
			if (g==0.0f){
				g2=hiprand_normal2(&localState);
				tmp.x=g2.x;
				tmp.y=g2.y;
				g2=hiprand_normal2(&localState);
				tmp.z=g2.x;
				g=g2.y;
			}else{
				tmp.x=g;
				g2=hiprand_normal2(&localState);
				tmp.y=g2.x;
				tmp.z=g2.y;
				g=0.0f;
			}
			surf2Dwrite(tmp,rand_buffer,16*j,i);
	    }
	    state[i] = localState;
	}
}

//
__global__ __launch_bounds__(ran_tpd) void refill_surface_taucd_gauss_rand (gpu_Ran *state, int n, int *count, bool SDCD_toggle){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	float4 tmp;
	float g=0.0f;
	float2 g2;
	if (i<n){
		int cnt=count[i];
		hiprandState localState = state[i];
	    for (int j=0; j<cnt;j++){
	    	tmp.x=hiprand_uniform (&localState);
			if (SDCD_toggle == true)
				tmp.w=d_tau_CD_f_t(tmp.x, d_At, d_Ct, d_Dt, d_tau_d_inv, d_g);
			else
				tmp.w=d_tau_CD_f_d_t(tmp.x, d_Adt, d_Bdt, d_Cdt, d_Ddt, d_tau_d_inv);
			if (g==0.0f){
				g2=hiprand_normal2(&localState);
				tmp.x=g2.x;
				tmp.y=g2.y;
				g2=hiprand_normal2(&localState);
				tmp.z=g2.x;
				g=g2.y;
			}else{
				tmp.x=g;
				g2=hiprand_normal2(&localState);
				tmp.y=g2.x;
				tmp.z=g2.y;
				g=0.0f;
			}
			surf2Dwrite(tmp,rand_buffer,16*j,i);
		}
		state[i] = localState;
	}
}

//
void gr_fill_surface_taucd_gauss_rand(gpu_Ran *gr, int sz, int count, bool SDCD_toggle, hipArray* d_taucd_gauss_rand){
	cudaBindSurfaceToArray(rand_buffer, d_taucd_gauss_rand);
    fill_surface_taucd_gauss_rand<<<(sz+ ran_tpd-1)/ ran_tpd, ran_tpd>>>(gr,sz,count,SDCD_toggle);
	CUT_CHECK_ERROR("kernel execution failed");
 	hipDeviceSynchronize();
}

//
void gr_refill_surface_taucd_gauss_rand(gpu_Ran *gr, int sz, int *count, bool SDCD_toggle, hipArray* d_taucd_gauss_rand){
	cudaBindSurfaceToArray(rand_buffer, d_taucd_gauss_rand);
	refill_surface_taucd_gauss_rand<<<(sz+ ran_tpd-1)/ ran_tpd, ran_tpd>>>(gr,sz,count,SDCD_toggle);
	CUT_CHECK_ERROR("kernel execution failed");
 	hipDeviceSynchronize();
}
