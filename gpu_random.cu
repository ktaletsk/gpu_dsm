#include "hip/hip_runtime.h"
// Copyright 2014 Marat Andreev
// 
// This file is part of gpu_dsm.
// 
// gpu_dsm is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// at your option) any later version.
// 
// gpu_dsm is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with gpu_dsm.  If not, see <http://www.gnu.org/licenses/>.

//  #include <hip/hip_runtime.h>
#include "gpu_random.h"
#include "random.h"
#include "pcd_tau.h"
#include "ensemble.h"
#include <fstream>

extern float step;
extern float mp,Mk;
extern int table_size;
extern float gamma_new_table_x[200000];

extern p_cd *pcd;
extern bool PD_flag;
// #include <iostream>

#if defined(_MSC_VER)
#define uint unsigned int
#endif

#include "cudautil.h"
#include "cuda_call.h"      
#include "textures_surfaces.h"

//CD constants
__constant__ float d_At, d_Ct, d_Dt, d_Adt, d_Bdt, d_Cdt, d_Ddt;
__constant__ float d_g, d_alpha, d_tau_0, d_tau_max, d_tau_d, d_tau_d_inv;
__constant__ bool d_PD_flag;

hipArray* d_gamma_table;
texture<float, hipTextureType1D, hipReadModeElementType> t_gamma_table;
__constant__ float d_step;
__constant__ float d_Mk, d_mp;
__device__ float d_p_At, d_p_Ct, d_p_Dt, d_p_g, d_p_Adt, d_p_Bdt, d_p_Cdt, d_p_Ddt, d_p_tau_d_inv; //Dynamic fdt parameters for given Nk in polydisperse solution
__constant__ float d_Be;

void gpu_ran_init () {
	cout << "preparing GPU random number generator parameters..\n";

	if(PD_flag){
		hipChannelFormatDesc channelDesc1 = hipCreateChannelDesc<float>();
		CUDA_SAFE_CALL(hipMallocArray(&d_gamma_table, &channelDesc1, table_size));
		CUDA_SAFE_CALL(hipMemcpyToArray(d_gamma_table, 0, 0, &gamma_new_table_x, table_size * sizeof(float), hipMemcpyHostToDevice));
		CUDA_SAFE_CALL(hipBindTextureToArray(t_gamma_table, d_gamma_table, channelDesc1));
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_step), &step, sizeof(float)));
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_mp), &mp, sizeof(float)));
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_Mk), &Mk, sizeof(float)));
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_Be), &Be, sizeof(float)));
	}
	else {
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_g), &(pcd->g), sizeof(float)));
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_alpha), &(pcd->alpha), sizeof(float)));
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tau_0), &(pcd->tau_0), sizeof(float)));
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tau_max), &(pcd->tau_max), sizeof(float)));
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tau_d), &(pcd->tau_d), sizeof(float)));
		float cdtemp = 1.0f / pcd->tau_d;
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_tau_d_inv), &(cdtemp), sizeof(float)));

		cdtemp = 1.0f / pcd->At;
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_At), &cdtemp, sizeof(float)));
		cdtemp = powf(pcd->tau_0, pcd->alpha);
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_Dt), &cdtemp, sizeof(float)));
		cdtemp = -1.0f / pcd->alpha;
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_Ct), &cdtemp, sizeof(float)));
		cdtemp = pcd->normdt / pcd->Adt;
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_Adt), &cdtemp, sizeof(float)));
		cdtemp = pcd->Bdt / pcd->normdt;
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_Bdt), &cdtemp, sizeof(float)));
		cdtemp = -1.0f / (pcd->alpha - 1.0f);
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_Cdt), &cdtemp, sizeof(float)));
		cdtemp = powf(pcd->tau_0, pcd->alpha - 1.0f);
		CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_Ddt), &(cdtemp), sizeof(float)));
	}

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_PD_flag), &PD_flag, sizeof(bool)));
	cout << "device random number generator parameters done\n";
}

//
__global__ __launch_bounds__(ran_tpd) void fill_surface_rand (gpu_Ran *state,int n,int count ){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	float tmp;
	if (i<n){
		hiprandState localState = state[i];
		for (int j=0; j<count;j++){
			tmp=hiprand_uniform (&localState);
			surf2Dwrite(tmp,rand_buffer,4*j,i);
		}
		state[i] = localState;
	}
}

//
__global__ __launch_bounds__(ran_tpd) void array_seed (gpu_Ran *gr,int sz,int seed_offset){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	if (i<sz) hiprand_init(seed_offset, i, 0, &gr[i]);
}

//
void gr_array_seed (gpu_Ran *gr,int sz, int seed_offset){      
	array_seed<<<(sz+ ran_tpd-1)/ ran_tpd, ran_tpd>>>(gr,sz, seed_offset);
	CUT_CHECK_ERROR("kernel execution failed");
 	hipDeviceSynchronize();
}

//
void gr_fill_surface_uniformrand(gpu_Ran *gr,int sz,int count , hipArray*  d_uniformrand){
	cudaBindSurfaceToArray(rand_buffer, d_uniformrand);
	CUT_CHECK_ERROR("kernel execution failed");
	fill_surface_rand<<<(sz+ ran_tpd-1)/ ran_tpd, ran_tpd>>>(gr,sz,count);
	CUT_CHECK_ERROR("kernel execution failed");
	hipDeviceSynchronize();
}

//lifetime generation from uniform random number p
__device__ __forceinline__ float d_tau_CD_f_d_t(float p, float d_Adt, float d_Bdt, float d_Cdt, float d_Ddt, float d_tau_d_inv) {
	return p < d_Bdt ? __powf(p * d_Adt + d_Ddt, d_Cdt) : d_tau_d_inv;
}
__device__ __forceinline__ float d_tau_CD_f_t(float p, float d_At, float d_Ct, float d_Dt, float d_tau_d_inv, float d_g) {
	return p < 1.0f - d_g ? __powf(p * d_At + d_Dt, d_Ct) : d_tau_d_inv;
}

__device__ void p_cd_(float Be, int Nk) {
	//Generates \tau_CD lifetimes
	//uses analytical approximation to P_cd parameters
	float At, Adt, Bdt, normdt;
	float g, alpha, tau_0, tau_max, tau_d;
	double z = (Nk + Be) / (Be + 1.0);
	g = 0.667f;
	if (Be != 1.0f) {
		//Analytical approximation to P_cd parameters for FSM
		//Unpublished Pilyugina E. (2012)

		alpha = (0.053f * logf(Be) + 0.31f) * powf(z, -0.012f * logf(Be) - 0.024f);
		tau_0 = 0.285f * powf(Be + 2.0f, 0.515f);
		tau_max = 0.025f * powf(Be + 2.0f, 2.6f) * powf(z, 2.83f);
		tau_d = 0.036f * powf(Be + 2.0f, 3.07f) * powf(z - 1.0f, 3.02f);
	} else {
		//Analytical approximation to P_cd parameters CFSM
		//Andreev, M., Feng, H., Yang, L., and Schieber, J. D.,J. Rheol. 58, 723 (2014).
		//DOI:10.1122/1.4869252

		alpha = 0.267096f - 0.375571f * expf(-0.0838237f * Nk);
		tau_0 = 0.460277f + 0.298913f * expf(-0.0705314f * Nk);
		tau_max = 0.0156137f * powf(float(Nk), 3.18849f);
		tau_d = 0.0740131f * powf(float(Nk), 3.18363f);
	}
	//init vars
	At = (1.0f - g) / (powf(tau_max, alpha) - powf(tau_0, alpha));
	Adt = (1.0f - g) * alpha / (alpha - 1.0f)
			/ (powf(tau_max, alpha) - powf(tau_0, alpha));
	Bdt = Adt * (powf(tau_max, alpha - 1.0f) - powf(tau_0, alpha - 1.0f));
	normdt = Bdt + g / tau_d;

	d_p_g=g;
	d_p_tau_d_inv = 1.0f / tau_d;
	d_p_At = 1.0f / At;
	d_p_Dt = powf(tau_0, alpha);
	d_p_Ct = -1.0f / alpha;
	d_p_Adt = normdt / Adt;
	d_p_Bdt = Bdt / normdt;
	d_p_Cdt = -1.0f / (alpha - 1.0f);
	d_p_Ddt = powf(tau_0, alpha - 1.0f);
}

//
__global__ __launch_bounds__(ran_tpd) void fill_surface_taucd_gauss_rand (gpu_Ran *state, int n, int count, bool SDCD_toggle){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	float4 tmp;
	float g=0.0f;
	float2 g2;
	if (i<n){
		hiprandState localState = state[i];
		for (int j=0; j<count;j++){
			//Pcd generation for new entanglements

			//Pick a uniform distributed random number
			tmp.x=hiprand_uniform (&localState);

			if (d_PD_flag){

				tmp.y=tex1D(t_gamma_table, hiprand_uniform(&localState)/d_step); //get molecular weight of background chain from table
				p_cd_(d_Be, (int)(tmp.y*d_mp/d_Mk + 0.5)); //Calculate pcd parameters
				if (SDCD_toggle == true)
					tmp.w = d_tau_CD_f_t(tmp.x, d_p_At, d_p_Ct, d_p_Dt, d_p_tau_d_inv, d_p_g);
				else
					tmp.w = d_tau_CD_f_d_t(tmp.x, d_p_Adt, d_p_Bdt, d_p_Cdt, d_p_Ddt, d_p_tau_d_inv);
			}
			else{
				if (SDCD_toggle == true)
					tmp.w = d_tau_CD_f_t(tmp.x, d_At, d_Ct, d_Dt, d_tau_d_inv, d_g);
				else
					tmp.w = d_tau_CD_f_d_t(tmp.x, d_Adt, d_Bdt, d_Cdt, d_Ddt, d_tau_d_inv);
			}

			//Q vector generation for new entanglements
			if (g==0.0f){
				g2=hiprand_normal2(&localState);
				tmp.x=g2.x;
				tmp.y=g2.y;
				g2=hiprand_normal2(&localState);
				tmp.z=g2.x;
				g=g2.y;
			}else{
				tmp.x=g;
				g2=hiprand_normal2(&localState);
				tmp.y=g2.x;
				tmp.z=g2.y;
				g=0.0f;
			}
			surf2Dwrite(tmp,rand_buffer,16*j,i);
	    }
	    state[i] = localState;
	}
}

//
__global__ __launch_bounds__(ran_tpd) void refill_surface_taucd_gauss_rand (gpu_Ran *state, int n, int *count, bool SDCD_toggle){
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	float4 tmp;
	float g=0.0f;
	float2 g2;
	if (i<n){
		int cnt=count[i];
		hiprandState localState = state[i];
	    for (int j=0; j<cnt;j++){
	    	tmp.x=hiprand_uniform (&localState);
			if (d_PD_flag){
				tmp.y=tex1D(t_gamma_table, hiprand_uniform(&localState)/d_step);
				p_cd_(d_Be, (int)(tmp.y*d_mp/d_Mk + 0.5));
				if (SDCD_toggle == true)
					tmp.w = d_tau_CD_f_t(tmp.x, d_p_At, d_p_Ct, d_p_Dt, d_p_tau_d_inv, d_p_g);
				else
					tmp.w = d_tau_CD_f_d_t(tmp.x, d_p_Adt, d_p_Bdt, d_p_Cdt, d_p_Ddt, d_p_tau_d_inv);
			}
			else{
				if (SDCD_toggle == true)
					tmp.w=d_tau_CD_f_t(tmp.x, d_At, d_Ct, d_Dt, d_tau_d_inv, d_g);
				else
					tmp.w=d_tau_CD_f_d_t(tmp.x, d_Adt, d_Bdt, d_Cdt, d_Ddt, d_tau_d_inv);
			}

			//Q vector generation for new entanglements
			if (g==0.0f){
				g2=hiprand_normal2(&localState);
				tmp.x=g2.x;
				tmp.y=g2.y;
				g2=hiprand_normal2(&localState);
				tmp.z=g2.x;
				g=g2.y;
			}else{
				tmp.x=g;
				g2=hiprand_normal2(&localState);
				tmp.y=g2.x;
				tmp.z=g2.y;
				g=0.0f;
			}
			surf2Dwrite(tmp,rand_buffer,16*j,i);
		}
		state[i] = localState;
	}
}

//
void gr_fill_surface_taucd_gauss_rand(gpu_Ran *gr, int sz, int count, bool SDCD_toggle, hipArray* d_taucd_gauss_rand){
	cudaBindSurfaceToArray(rand_buffer, d_taucd_gauss_rand);
    fill_surface_taucd_gauss_rand<<<(sz+ ran_tpd-1)/ ran_tpd, ran_tpd>>>(gr,sz,count,SDCD_toggle);
	CUT_CHECK_ERROR("kernel execution failed");
 	hipDeviceSynchronize();
}

//
void gr_refill_surface_taucd_gauss_rand(gpu_Ran *gr, int sz, int *count, bool SDCD_toggle, hipArray* d_taucd_gauss_rand){
	cudaBindSurfaceToArray(rand_buffer, d_taucd_gauss_rand);
	refill_surface_taucd_gauss_rand<<<(sz+ ran_tpd-1)/ ran_tpd, ran_tpd>>>(gr,sz,count,SDCD_toggle);
	CUT_CHECK_ERROR("kernel execution failed");
 	hipDeviceSynchronize();
}
