
#include <hip/hip_runtime.h>
#include <iostream>

int gpu_check() {
    int* count = new int;
    hipError_t x = hipGetDeviceCount(count);
    if (x==hipSuccess) return 0;
    else if (x==hipErrorNoDevice) return -1;
    else if (x==hipErrorInsufficientDriver) return -2;
    return 1;
}
