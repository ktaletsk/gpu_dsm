#include "hip/hip_runtime.h"
// Copyright 2014 Marat Andreev
// 
// This file is part of gpu_dsm.
// 
// gpu_dsm is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// at your option) any later version.
// 
// gpu_dsm is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
// 
// You should have received a copy of the GNU General Public License
// along with gpu_dsm.  If not, see <http://www.gnu.org/licenses/>.

//Equilibrium versions of kernels
//flow deformation turn off
//EQ_chain_CD_kernel fills s_correlator with of diagonal stress component

//      correlator constant
__constant__ int d_correlator_res;

//entanglement parallel part of the code
//2D kernel: i- entanglement index j - chain index
__global__ __launch_bounds__(tpb_strent_kernel*tpb_strent_kernel) void EQ_strent_kernel(chain_head* gpu_chain_heads, int *d_offset, float4 *d_new_strent,float *d_new_tau_CD) {
	//Calculate kernel index
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	//Check if kernel index is outside boundaries
	if ((j >= dn_cha_per_call) || (i >= d_z_max))
		return;

	int tz = gpu_chain_heads[j].Z; //Current chain size
	if (i >= tz) //Check if entaglement index is over chain size
		return;

	//When new entaglements are created we need to shift index +1(destruction, skip one strent), 0(nothing happens) or -1(new strent created before)
	int oft = d_offset[j]; //Offset for current chain

	//fetch
	float4 QN;
	if (fetch_new_strent(i, oft))
		QN = d_new_strent[j]; //second check if strent created last time step should go here
	else
		QN = tex2D(t_a_QN, make_offset(i, oft), j); // all access to strents is done through two operations: first texture fetch
	float tcd;
	if (dCD_flag) {//If constraint dynamics is enabled
		if (fetch_new_strent(i, oft))
			tcd = d_new_tau_CD[j];
		else
			tcd = tex2D(t_a_tCD, make_offset(i, oft), j);
	} else
		tcd = 0;

	//write
	surf2Dwrite(QN, s_b_QN, 16 * i, j);
	surf2Dwrite(tcd, s_b_tCD, 4 * i, j);

	//fetch next strent (if strent is not the last)
	if (i < tz - 1) {
		float2 wsh = make_float2(0.0f, 0.0f); //Variable for shifting probability
		float4 QN2; //Q for next strent
		if (fetch_new_strent(i + 1, oft))
			QN2 = d_new_strent[j];
		else
			QN2 = tex2D(t_a_QN, make_offset(i + 1, oft), j);

		//w_shift probability calc
		float Q  = QN.x  * QN.x  + QN.y  * QN.y  + QN.z  * QN.z;
		float Q2 = QN2.x * QN2.x + QN2.y * QN2.y + QN2.z * QN2.z;

		//Calculating probability of 1 Kuhn step to jump from i strand to (i+1) strand
		if (QN2.w > 1.0f) { //N=1 mean that shift is not possible, also it will lead to dividing by zero
			//float prefact=__powf( __fdividef(QN.w*QN2.w,(QN.w+1)*(QN2.w-1)),0.75f);

			float sig1 = __fdividef(0.75f, QN.w * (QN.w + 1));
			float sig2 = __fdividef(0.75f, QN2.w * (QN2.w - 1));
			float prefact1 = (Q == 0.0f) ? 1.0f : __fdividef(QN.w, (QN.w + 1));
			float prefact2 = (Q2 == 0.0f) ? 1.0f : __fdividef(QN2.w, (QN2.w - 1));
			float f1 = (Q == 0.0f) ? 2.0f * QN.w + 0.5f : QN.w;
			float f2 = (Q2 == 0.0f) ? 2.0f * QN2.w - 0.5f : QN2.w;
			float friction = __fdividef(2.0f, f1 + f2);
			wsh.x = friction * __powf(prefact1 * prefact2, 0.75f)* __expf(Q * sig1 - Q2 * sig2);
		}
		if (QN.w > 1.0f) {//N=1 mean that shift is not possible, also it will lead to dividing by zero

			float sig1 = __fdividef(0.75f, QN.w * (QN.w - 1.0f));
			float sig2 = __fdividef(0.75f, QN2.w * (QN2.w + 1.0f));
			float prefact1 = (Q == 0.0f) ? 1.0f : __fdividef(QN.w, (QN.w - 1.0f));
			float prefact2 = (Q2 == 0.0f) ? 1.0f : __fdividef(QN2.w, (QN2.w + 1.0f));
			float f1 = (Q == 0.0f) ? 2.0f * QN.w - 0.5f : QN.w;
			float f2 = (Q2 == 0.0f) ? 2.0f * QN2.w + 0.5f : QN2.w;
			float friction = __fdividef(2.0f, f1 + f2);
			wsh.y = friction * __powf(prefact1 * prefact2, 0.75f)* __expf(-Q * sig1 + Q2 * sig2);
		}
// 	    surf2Dwrite(wsh.x,s_W_SD_pm,8*i,j);//TODO funny bug i have no idea but doesn't work other way
// 	    surf2Dwrite(wsh.y,s_W_SD_pm,8*i+4,j);//seems to work with float4 below
		surf2Dwrite(wsh.x + wsh.y + dCD_flag * (tcd + d_CD_create_prefact * (QN.w - 1.0f)), s_sum_W, 4 * i, j);
	}
}

__global__ __launch_bounds__(tpb_chain_kernel) void EQ_chain_kernel(chain_head* gpu_chain_heads, float *tdt, float *reach_flag, float next_sync_time, int *d_offset, float4 *d_new_strent, float *d_new_tau_CD, int *d_correlator_time, int *rand_used, int *tau_CD_used) {
	int i = blockIdx.x * blockDim.x + threadIdx.x; //Chain index

	if (i >= dn_cha_per_call)
		return;

	//setup local variables
	int tz = gpu_chain_heads[i].Z;
	uint oft = d_offset[i];
	d_offset[i] = offset_code(0xffff, +1);

	if (reach_flag[i]!=0) {
		return;
	}
	if (((gpu_chain_heads[i].time >= next_sync_time)&&
	     (d_universal_time+next_sync_time<=d_correlator_time[i] * d_correlator_res)) ||  (gpu_chain_heads[i].stall_flag != 0))
	   {
		reach_flag[i] = 1;
		gpu_chain_heads[i].time-=next_sync_time;
		tdt[i] = 0.0f;
		return;
	}
	float4 new_strent = d_new_strent[i];
	float new_tCD = d_new_tau_CD[i];

	//check for correlator
	if (d_universal_time+gpu_chain_heads[i].time > d_correlator_time[i] * d_correlator_res) {//TODO add d_correlator_time to gpu_chain_heads
		float4 sum_stress = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
		for (int j = 0; j < tz; j++) {
			float4 QN1;
			if (fetch_new_strent(j, oft))
				QN1 = new_strent;
			else
				QN1 = tex2D(t_a_QN, make_offset(j, oft), i);
			sum_stress.x -= __fdividef(3.0f * QN1.x * QN1.y, QN1.w);
			sum_stress.y -= __fdividef(3.0f * QN1.y * QN1.z, QN1.w);
			sum_stress.z -= __fdividef(3.0f * QN1.x * QN1.z, QN1.w);
		}
		surf2Dwrite(sum_stress, s_correlator, 16 * d_correlator_time[i], i);
		d_correlator_time[i]++;
		if (d_universal_time+gpu_chain_heads[i].time > d_correlator_time[i] * d_correlator_res){
		  return;
		  //do nothing until next step
		}
	}

	// sum W_SD_shifts
	float sum_wshpm = 0.0f;
	float tsumw;
	for (int j = 0; j < tz - 1; j++) {
		surf2Dread(&tsumw, s_sum_W, 4 * j, i);
		sum_wshpm += tsumw;
	}

	// W_SD_c/d calc
	float W_SD_c_1 = 0.0f, W_SD_d_1 = 0.0f;
	float W_SD_c_z = 0.0f, W_SD_d_z = 0.0f;
	//declare vars to reuse later
	float4 QNheadn, QNtailp;

	// first strent
	float4 QNhead;
	if (fetch_new_strent(0, oft))
		QNhead = new_strent;
	else
		QNhead = tex2D(t_a_QN, make_offset(0, oft), i);

	//last strent
	float4 QNtail;
	if (fetch_new_strent(tz - 1, oft))
		QNtail = new_strent;
	else
		QNtail = tex2D(t_a_QN, make_offset(tz - 1, oft), i);

	float W_CD_c_z = dCD_flag * d_CD_create_prefact * (QNtail.w - 1.0f); //Create CD on the last strand

	if (tz == 1) {
		W_SD_c_1 = __fdividef(1.0f, (dBe * dnk));
		W_SD_c_z = W_SD_c_1;
	} else {
		if (QNhead.w == 1.0f) {
			//destruction
			if (fetch_new_strent(1, oft))
				QNheadn = new_strent;
			else
				QNheadn = tex2D(t_a_QN, make_offset(1, oft), i);
			float f2 = (tz == 2) ? QNheadn.w + 0.25f : 0.5f * QNheadn.w;
			W_SD_d_1 = __fdividef(1.0f, 0.75f + f2);
		} else {
			//creation
			W_SD_c_1 = __fdividef(2.0f, dBe * (QNhead.w + 0.5f));
		}

		if (QNtail.w == 1.0f) {
			//destruction
			if (fetch_new_strent(tz - 2, oft))
				QNtailp = new_strent;
			else
				QNtailp = tex2D(t_a_QN, make_offset(tz - 2, oft), i);
			float f1 = (tz == 2) ? QNtailp.w + 0.25f : 0.5f * QNtailp.w;
			W_SD_d_z = __fdividef(1.0f, 0.75f + f1);
		} else {
			//creation
			W_SD_c_z = __fdividef(2.0f, dBe * (QNtail.w + 0.5f));
		}
	}

	float sumW = sum_wshpm + W_SD_c_1 + W_SD_c_z + W_SD_d_1 + W_SD_d_z + W_CD_c_z;
	tdt[i] = __fdividef(1.0f, sumW);
	if (tdt[i] == 0.0f)
		gpu_chain_heads[i].stall_flag = 1;
	if (isnan(tdt[i]))
		gpu_chain_heads[i].stall_flag = 2;
	if (isinf(tdt[i]))
		gpu_chain_heads[i].stall_flag = 3;
	gpu_chain_heads[i].time += tdt[i];
// 	surf2Dread(&tdt[i],rand_buffer,4*0,i);

	float pr = (sumW) * tex2D(t_uniformrand, rand_used[i], i);
	rand_used[i]++;		//TODO just use step count constant instead of rand used
	int j = 0;
	float tpr = 0.0f;
	if (tz != 1)
		surf2Dread(&tpr, s_sum_W, 4 * j, i);

	// picking where(which strent) jump process will happen
	// excluding SD creation destruction
	// perhaps one of the most time consuming parts of the code
	while ((pr >= tpr) && (j < tz - 2)) {
		pr -= tpr;
		j++;
		surf2Dread(&tpr, s_sum_W, 4 * j, i);

	}

// 	  for (int j=0;j<tz-1;j++)
	if (pr < tpr) {
		// ok we pick some  strent j
		// now we need to decide which(SD shift or CDd CDc) jump process will happen
		// TODO check if order will have an effect on performance

		float4 QN1 = tex2D(t_a_QN, make_offset(j, oft), i);
		if (fetch_new_strent(j, oft))
			QN1 = new_strent;
		float4 QN2 = tex2D(t_a_QN, make_offset(j + 1, oft), i);
		if (fetch_new_strent(j + 1, oft))
			QN2 = new_strent;

		// 1. CDd (destruction by constraint dynamics)

		float wcdd;
		if (dCD_flag) {
			wcdd = tex2D(t_a_tCD, make_offset(j, oft), i); //Read CD
			if (fetch_new_strent(j, oft))
				wcdd = new_tCD;
		} else
			wcdd = 0;
		if (pr < wcdd) {

			float4 temp = make_float4(QN1.x + QN2.x, QN1.y + QN2.y,
					QN1.z + QN2.z, QN1.w + QN2.w);
			if ((j == tz - 2) || (j == 0)) {
				temp = make_float4(0.0f, 0.0f, 0.0f, QN1.w + QN2.w);
			}
			surf2Dwrite(temp, s_b_QN, 16 * (j + 1), i);
			d_offset[i] = offset_code(j, +1);
			gpu_chain_heads[i].Z--;

			return;
		} else {
			pr -= wcdd;
		}

		// 2. SD shift

		// SD shift probs are not saved from entanglement parallel part
		// so we need to recalculate it
		float2 twsh = make_float2(0.0f, 0.0f);
		float Q = QN1.x * QN1.x + QN1.y * QN1.y + QN1.z * QN1.z;
		float Q2 = QN2.x * QN2.x + QN2.y * QN2.y + QN2.z * QN2.z;

		if (QN2.w > 1.0f) {	//N=1 mean that shift is not possible, also ot will lead to dividing on zero error
			//float prefact=__powf( __fdividef(QN1.w*QN2.w,(QN1.w+1)*(QN2.w-1)),0.75f);
			//TODO replace powf with sqrt(x*x*x)

			float sig1 = __fdividef(0.75f, QN1.w * (QN1.w + 1));
			float sig2 = __fdividef(0.75f, QN2.w * (QN2.w - 1));
			float prefact1 = (Q == 0.0f) ? 1.0f : __fdividef(QN1.w, (QN1.w + 1));
			float prefact2 = (Q2 == 0.0f) ? 1.0f : __fdividef(QN2.w, (QN2.w - 1));
			float f1 = (Q == 0.0f) ? 2.0f * QN1.w + 0.5f : QN1.w;
			float f2 = (Q2 == 0.0f) ? 2.0f * QN2.w - 0.5f : QN2.w;
			float friction = __fdividef(2.0f, f1 + f2);
			twsh.x = friction * __powf(prefact1 * prefact2, 0.75f) * __expf(Q * sig1 - Q2 * sig2);
		}
		if (QN1.w > 1.0f) {	//N=1 mean that shift is not possible, also ot will lead to dividing on zero error

			float sig1 = __fdividef(0.75f, QN1.w * (QN1.w - 1.0f));
			float sig2 = __fdividef(0.75f, QN2.w * (QN2.w + 1.0f));
			float prefact1 = (Q == 0.0f) ? 1.0f : __fdividef(QN1.w, (QN1.w - 1.0f));
			float prefact2 = (Q2 == 0.0f) ? 1.0f : __fdividef(QN2.w, (QN2.w + 1.0f));
			float f1 = (Q == 0.0f) ? 2.0f * QN1.w - 0.5f : QN1.w;
			float f2 = (Q2 == 0.0f) ? 2.0f * QN2.w + 0.5f : QN2.w;
			float friction = __fdividef(2.0f, f1 + f2);
			twsh.y = friction * __powf(prefact1 * prefact2, 0.75f) * __expf(-Q * sig1 + Q2 * sig2);
		}

		if (pr < twsh.x + twsh.y) {

			if (pr < twsh.x) {
				QN1.w = QN1.w + 1;
				QN2.w = QN2.w - 1;
			} else {
				QN1.w = QN1.w - 1;
				QN2.w = QN2.w + 1;
			}
			surf2Dwrite(QN1, s_b_QN, 16 * j, i);
			surf2Dwrite(QN2, s_b_QN, 16 * (j + 1), i);
			return;
		} else {
			pr -= twsh.x + twsh.y;
		}

		// 3. CDc (creation by constraint dynamics in the middle)
		float wcdc = dCD_flag * d_CD_create_prefact * (QN1.w - 1.0f); //
		if (pr < wcdc) {
			if (tz == d_z_max)
				return;		// possible detail balance issue
			float4 temp = tex2D(t_taucd_gauss_rand, tau_CD_used[i], i);
			tau_CD_used[i]++;
			gpu_chain_heads[i].Z++;
			d_new_tau_CD[i] = d_tau_CD_f_d_t(temp.w);//__fdividef(1.0f,d_tau_d);
			float newn = floorf(0.5f + __fdividef(pr * (QN1.w - 2.0f), wcdc)) + 1.0f;
			if (j == 0) {
				temp.w = QN1.w - newn;
				float sigma = __fsqrt_rn(__fdividef(temp.w, 3.0f));
				temp.x *= sigma;
				temp.y *= sigma;
				temp.z *= sigma;
				surf2Dwrite(temp, s_b_QN, 16 * 0, i);
				d_offset[i] = offset_code(0, -1);
				d_new_strent[i] = make_float4(0.0f, 0.0f, 0.0f, newn);
				return;
			}
			temp.w = newn;
			float sigma = __fsqrt_rn(__fdividef(newn * (QN1.w - newn), 3.0f * QN1.w));
			float ration = __fdividef(newn, QN1.w);
			temp.x *= sigma;
			temp.y *= sigma;
			temp.z *= sigma;
			temp.x += QN1.x * ration;
			temp.y += QN1.y * ration;
			temp.z += QN1.z * ration;
			surf2Dwrite(
					make_float4(QN1.x - temp.x, QN1.y - temp.y, QN1.z - temp.z,
							QN1.w - newn), s_b_QN, 16 * j, i);
			d_offset[i] = offset_code(j, -1);
			d_new_strent[i] = temp;
			return;
		} else {
			pr -= wcdc;
		}
	} else {
		pr -= tpr;
	}

	//None of the processes in the middle of the chain happened
	//Now check processes on the left end

	// 4. w_CD_c_z (creation by constraint dynamics on the left end)
	if (pr < W_CD_c_z) {
		if (tz == d_z_max)
			return;	// possible detail balance issue

		float4 temp = tex2D(t_taucd_gauss_rand, tau_CD_used[i], i);
		tau_CD_used[i]++;
		gpu_chain_heads[i].Z++;
		d_new_tau_CD[i] = d_tau_CD_f_d_t(temp.w);	//__fdividef(1.0f,d_tau_d);

		float newn = 1.0f + floorf(0.5f + __fdividef(pr * (QNtail.w - 2.0f), W_CD_c_z));
//		floorf(__fdividef(pr*(QNtail.w-1.0f),W_CD_c_z))+1.0f;
// 	    gpu_chain_heads[i].dummy=1.0f+__fdiv_rn(pr*(QNtail.w-2.0f),W_CD_c_z);

		temp.w = newn;
		float sigma = (tz == 1) ? 0.0f : __fsqrt_rn(__fdividef(temp.w, 3.0f));
		temp.x *= sigma;
		temp.y *= sigma;
		temp.z *= sigma;
		surf2Dwrite(make_float4(0.0f, 0.0f, 0.0f, QNtail.w - newn), s_b_QN, 16 * (tz - 1), i);
		d_offset[i] = offset_code(tz - 1, -1);
		d_new_strent[i] = temp;
		return;
	} else {
		pr -= W_CD_c_z;
	}

	// 5. w_SD_c (creation by sliding dynamics)

	if (pr < W_SD_c_1 + W_SD_c_z) {
		if (tz == d_z_max)
			return;	// possible detail balance issue
		float4 temp = tex2D(t_taucd_gauss_rand, tau_CD_used[i], i);
		tau_CD_used[i]++;
		gpu_chain_heads[i].Z++;
//		d_new_tau_CD[i]=__fdividef(1.0f,d_tau_d);
		d_new_tau_CD[i] = d_tau_CD_f_t(temp.w);

		if (pr < W_SD_c_1) {
			temp.w = QNhead.w - 1.0f;
			float sigma = (tz == 1) ? 0.0f : __fsqrt_rn(__fdividef(temp.w, 3.0f));
			temp.x *= sigma;
			temp.y *= sigma;
			temp.z *= sigma;
			surf2Dwrite(temp, s_b_QN, 16 * 0, i);//TODO maybe deformation should be applied here
			d_offset[i] = offset_code(0, -1);
			d_new_strent[i] = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
		} else {
			temp.w = QNtail.w - 1.0f;
			float sigma = (tz == 1) ? 0.0f : __fsqrt_rn(__fdividef(temp.w, 3.0f));
			temp.x *= sigma;
			temp.y *= sigma;
			temp.z *= sigma;
			surf2Dwrite(make_float4(0.0f, 0.0f, 0.0f, 1.0f), s_b_QN, 16 * (tz - 1), i);//TODO maybe deformation should be applied here
			d_offset[i] = offset_code(tz - 1, -1);
			d_new_strent[i] = temp;
		}

		return;
	} else {
		pr -= W_SD_c_1 + W_SD_c_z;

	}

	// 6. Destruction by constraint dynamics
	if (pr < W_SD_d_1 + W_SD_d_z) {	//to delete entanglement
	// update cell and neigbours
	//clear W_sd
	//
	//form a list of free cell
		gpu_chain_heads[i].Z--;
		if (pr < W_SD_d_1) {
			surf2Dwrite(make_float4(0.0f, 0.0f, 0.0f, QNheadn.w + 1.0f), s_b_QN,
					16 * 1, i);
			d_offset[i] = offset_code(0, +1);
		} else {
			surf2Dwrite(make_float4(0.0f, 0.0f, 0.0f, QNtailp.w + 1.0f), s_b_QN,
					16 * (tz - 2), i);
			d_offset[i] = offset_code(tz, +1);

		}
		return;

	} else {
		pr -= W_SD_d_1 + W_SD_d_z;
	}

}

