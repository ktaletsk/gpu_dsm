#include <iostream>
#if defined(_MSC_VER)
    #include <windows.h>
#endif
#include <stdio.h>
#include <cuda_gl_interop.h>

#include "cudautil.h"

    void checkCUDA(int dev)
    {
    //checks for presence of GPU
    //selects device number dev to use
    //just copy of "CUDA programming guide" code
	int deviceCount;
	CUDA_SAFE_CALL_NOSYNC(hipGetDeviceCount(&deviceCount));
	if (deviceCount == 0) {
		fprintf(stderr, "no CUDA device found!\n");
		exit(1);
	}


// 	int dev = deviceCount - 1;
	printf("device Count %d \n", deviceCount);
	if (dev>=deviceCount) printf("device %d not present\n", dev);
	hipDeviceProp_t deviceProp;
	CUDA_SAFE_CALL_NOSYNC(hipGetDeviceProperties(&deviceProp, dev));
	if (deviceProp.major < 1) {
		fprintf(stderr, "device %d does not support CUDA!\n", dev);
		exit(1);
	}

	printf("Using device %d: %s\n", dev, deviceProp.name );
	CUDA_SAFE_CALL(hipSetDevice(dev));
	CUDA_SAFE_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
    }


